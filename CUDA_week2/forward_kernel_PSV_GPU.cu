#include "hip/hip_runtime.h"

/*
* Created by: Min Basnet
* 2020.April.16
* Kathmandu, Nepal
*/
#include <time.h>
#include <iostream>
#include <fstream>
#include <cmath>
#include <vector>

#include "globvar.cuh"
#include "util.cu"
#include "fd_cpml.cuh"
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}



__global__ void kernel_zero(int tf, int nt, int fwi_z1, int fwi_z2, int fwi_x1, int fwi_x2, int fwi_dz, int fwi_dx, int nft, int nzt,int nxt, int nfx, real_sim* fwi_sxx, real_sim* fwi_szx, real_sim* fwi_szz, real_sim* fwi_vx, real_sim*
    fwi_vz, real_sim* sxx, real_sim* szx, real_sim* szz, real_sim* vx, real_sim* vz) {


    //(int tf, int nt, int fwi_z1, int fwi_z2, int fwi_x1, int fwi_x2, int fwi_dz, int fwi_dx, int nft, int nzt, int nxt,nfx real_sim* fwi_sxx, real_sim* fwi_szx, real_sim* fwi_szz, real_sim* fwi_vx, real_sim*
   // fwi_vz, real_sim* sxx, real_sim* szx, real_sim* szz, real_sim* vx, real_sim* vz)


    int iz = blockIdx.x * blockDim.x + threadIdx.x;
    int ix = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (iz >= fwi_z1 && iz < fwi_z2 && ((iz % fwi_dz) == fwi_z1 % fwi_dz) && ix >= fwi_x1 && ix < fwi_x2 && ((ix % fwi_dx) == fwi_x1 % fwi_dx)) { // storing only a portion and with grid inteval
        int zf = (iz - fwi_z1) / fwi_dz; // z index for fwi gradient storage
    
                                         // printf("Hello Executed below pz \n");
        
            int xf = (ix - fwi_x1) / fwi_dx; // x index for fwi gradient storage
            
            int offset = tf * nzt * nxt + zf * nxt + xf;
            fwi_sxx[offset] = sxx[iz * nxt + ix];

            fwi_szx[offset] = szx[iz * nxt + ix];
            fwi_szz[offset] = szz[iz * nxt + ix];

            fwi_vx[offset] = vx[iz * nxt + ix];
            fwi_vz[offset] = vz[iz * nxt + ix];

        
    }
    else {
        return;
    }
}

__global__ void kernel_one(int ishot, int nt, int nzt, int nxt, int fpad, int ppad, real_sim dt, real_sim dx, real_sim dz,
    int fdorder, real_sim* vx, real_sim* vz, real_sim* sxx,
    real_sim* szx, real_sim* szz, real_sim* lam, real_sim* mu,
    real_sim* mu_zx, real_sim* rho_zp, real_sim* rho_xp, int npml,
    real_sim* a, real_sim* b, real_sim* K, real_sim* a_half, real_sim* b_half, real_sim* K_half,
    real_sim* mem_vx_x, real_sim* mem_vx_z, real_sim* mem_vz_x, real_sim* mem_vz_z,
    real_sim* mem_sxx_x, real_sim* mem_szx_x, real_sim* mem_szz_z, real_sim* mem_szx_z, bool fsurf) {

    //**********************************************************************************
    real_sim sxx_x, szx_x, szx_z, szz_z; // spatial stress derivatives
    real_sim vx_x, vx_z, vz_x, vz_z; // spatial velocity derivatives
    int nz1, nz2, nx1, nx2; // The computational grid boundaries
    int px, pz; // index for PML arrys
    int isnap; // to take snapshots for data storage

    int tf, zf, xf; // Index parameters for fwi data storage

    real_sim hc[2] = { 1.0, 1.0 };

    // Initial calculation of indices
    //---------------------------------------------
    nz1 = fpad; nz2 = nzt - fpad; nx1 = fpad; nx2 = nxt - fpad; // index variables

    // index to map PML at the right or positive end
    int pnx, pnz;
    pnx = nxt - 2 * ppad + fpad - 1; // nx + ppad + npml + 1 and nx = nxt - 2*ppad

    if (fsurf) {
        pnz = nzt - 2 * ppad - 1; // nz + ppad + npml + 1 and nz = nzt - ppad - fpad
    }
    else {
        pnz = nzt - 2 * ppad + fpad - 1; // nz + ppad + npml + 1 and nz = nzt - 2*ppad
    }

    real_sim dxi = 1.0 / dx; real_sim dzi = 1.0 / dz; // inverse of dx and dz
//**********************************************************************************************************


    int iz = blockIdx.x * blockDim.x + threadIdx.x;
    int ix = blockIdx.y * blockDim.y + threadIdx.y;


    switch (fdorder) {
    case(2):
        if (ix < nx2 && ix >= nx1 && iz >= nz1 && iz < nz2)

        {

            // Calculate spatial velocity derivatives
            vx_x = dxi * hc[1] * (vx[iz * nxt + ix] - vx[iz * nxt + (ix - 1)]);
            vz_x = dxi * hc[1] * (vz[iz * nxt + (ix + 1)] - vz[iz * nxt + ix]);
            vx_z = dzi * hc[1] * (vx[(iz + 1) * nxt + ix] - vx[iz * nxt + ix]);
            vz_z = dzi * hc[1] * (vz[iz * nxt + ix] - vz[(iz - 1) * nxt + ix]);
            //************************************************************************************************
    // ---------------------------------------------------
    // CPML layers for stress tensor kernel
    // ---------------------------------------------------




        // ---------------------------------------------------
        // CPML layers for stress tensor kernel
         //---------------------------------------------------

            if (npml > 0) {

                if (ix >= fpad && ix <= ppad)
                { // left CPML
                  // Mapping the static CPML and memory variables to
                    px = ix - fpad; // the memory array index


                    mem_vx_x[iz * 2 * (npml + 1) + px] = b[px] * mem_vx_x[iz * 2 * (npml + 1) + px] + a[px] * vx_x;
                    mem_vz_x[iz * 2 * (npml + 1) + px] = b_half[px] * mem_vz_x[iz * 2 * (npml + 1) + px] + a_half[px] * vz_x;

                    vx_x = vx_x / K[px] + mem_vx_x[iz * 2 * (npml + 1) + px];
                    vz_x = vz_x / K_half[px] + mem_vz_x[iz * 2 * (npml + 1) + px];

                } // cpml left



                if (ix >= (nxt - ppad - 1) && ix < nxt - fpad) { // right CPML
                  // Mapping the static CPML and memory variables to

                    px = ix - pnx; // The PML factors index

           //         
                    mem_vx_x[iz * 2 * (npml + 1) + px] = b[px] * mem_vx_x[iz * 2 * (npml + 1) + px] + a[px] * vx_x;
                    mem_vz_x[iz * 2 * (npml + 1) + px] = b_half[px] * mem_vz_x[iz * 2 * (npml + 1) + px] + a_half[px] * vz_x;

                    vx_x = vx_x / K[px] + mem_vx_x[iz * 2 * (npml + 1) + px];
                    vz_x = vz_x / K_half[px] + mem_vz_x[iz * 2 * (npml + 1) + px];



                } // cpml right


                if (iz >= fpad && iz <= ppad && !fsurf) { // top CPML
                  // Mapping the static CPML and memory variables to
                    pz = iz - fpad; // the memory array index


                    mem_vz_z[pz * nxt + ix] = b[pz] * mem_vz_z[pz * nxt + ix] + a[pz] * vz_z;
                    mem_vx_z[pz * nxt + ix] = b_half[pz] * mem_vx_z[pz * nxt + ix] + a_half[pz] * vx_z;

                    vz_z = vz_z / K[pz] + mem_vz_z[pz * nxt + ix];
                    vx_z = vx_z / K_half[pz] + mem_vx_z[pz * nxt + ix];

                    //std::cout << pz<< ", ";

                } // cpml top



                if (iz >= (nzt - ppad - 1) && iz < nzt - fpad) { // bottom CPML
                  // Mapping the static CPML and memory variables to
                    pz = iz - pnz; // The PML factors index
                    mem_vz_z[pz * nxt + ix] = b[pz] * mem_vz_z[pz * nxt + ix] + a[pz] * vz_z;
                    mem_vx_z[pz * nxt + ix] = b_half[pz] * mem_vx_z[pz * nxt + ix] + a_half[pz] * vx_z;

                    vz_z = vz_z / K[pz] + mem_vz_z[pz * nxt + ix];
                    vx_z = vx_z / K_half[pz] + mem_vx_z[pz * nxt + ix];
                    //std::cout << pz<< ", ";

                } // cpml bottom

            }    // npml>0
            __syncthreads();

            ////// updating stresses

            szx[iz * nxt + ix] += dt * mu_zx[iz * (nxt - 1) + ix] * (vz_x + vx_z);
            sxx[iz * nxt + ix] += dt * (lam[iz * nxt + ix] * (vx_x + vz_z) + (2.0f * mu[iz * nxt + ix] * vx_x));
            szz[iz * nxt + ix] += dt * (lam[iz * nxt + ix] * (vx_x + vz_z) + (2.0f * mu[iz * nxt + ix] * vz_z));

            // Override stress for free surface implementation

            if (fsurf && iz == fpad) {
                // Free surface at z = 0 or nz = fpad

                // Denise manual  page 13
                szz[fpad * nxt + ix] = 0.0;
                szx[fpad * nxt + ix] = 0.0;
                sxx[fpad * nxt + ix] = 4.0 * dt * vx_x * (lam[fpad * nxt + ix] * mu[fpad * nxt + ix] + mu[fpad * nxt + ix] * mu[fpad * nxt + ix])
                    / (lam[fpad * nxt + ix] + 2.0 * mu[fpad * nxt + ix]);



            }


        }
        else {
            return;
        }

    default:
        //  std::cout << "FDORDER = " << fdorder << npml << std::endl;
        //  std::cout << "FD order mismatch. <SIMULATION ABORTED>" << std::endl;
          //exit(0);
       // printf("FD order mismatch. <SIMULATION ABORTED>\n");
    } // end of switch


}

__global__ void kernel_two(int ishot, int nt, int nzt, int nxt, int fpad, int ppad, real_sim dt, real_sim dx, real_sim dz,
    int fdorder, real_sim* vx, real_sim* vz, real_sim* sxx,
    real_sim* szx, real_sim* szz, real_sim* lam, real_sim* mu,
    real_sim* mu_zx, real_sim* rho_zp, real_sim* rho_xp, int npml,
    real_sim* a, real_sim* b, real_sim* K, real_sim* a_half, real_sim* b_half, real_sim* K_half,
    real_sim* mem_vx_x, real_sim* mem_vx_z, real_sim* mem_vz_x, real_sim* mem_vz_z,
    real_sim* mem_sxx_x, real_sim* mem_szx_x, real_sim* mem_szz_z, real_sim* mem_szx_z, bool fsurf) {


    real_sim sxx_x, szx_x, szx_z, szz_z; // spatial stress derivatives
    real_sim vx_x, vx_z, vz_x, vz_z; // spatial velocity derivatives
    int nz1, nz2, nx1, nx2; // The computational grid boundaries
    int px, pz; // index for PML arrys
    int isnap; // to take snapshots for data storage

    int tf, zf, xf; // Index parameters for fwi data storage


    real_sim hc[2] = { 1.0, 1.0 };

    // Initial calculation of indices
    //---------------------------------------------
    nz1 = fpad; nz2 = nzt - fpad; nx1 = fpad; nx2 = nxt - fpad; // index variables

    // index to map PML at the right or positive end
    int pnx, pnz;
    pnx = nxt - 2 * ppad + fpad - 1; // nx + ppad + npml + 1 and nx = nxt - 2*ppad

    if (fsurf) {
        pnz = nzt - 2 * ppad - 1; // nz + ppad + npml + 1 and nz = nzt - ppad - fpad
    }
    else {
        pnz = nzt - 2 * ppad + fpad - 1; // nz + ppad + npml + 1 and nz = nzt - 2*ppad
    }

    real_sim dxi = 1.0 / dx; real_sim dzi = 1.0 / dz; // inverse of dx and dz


    //*************************************************

    int iz = blockIdx.x * blockDim.x + threadIdx.x;
    int ix = blockIdx.y * blockDim.y + threadIdx.y;


    switch (fdorder) {
    case(2):
        if (ix < nx2 && ix >= nx1 && iz >= nz1 && iz < nz2)
        {
            sxx_x = dxi * hc[1] * (sxx[iz * nxt + ix + 1] - sxx[iz * nxt + ix]);
            szx_z = dxi * hc[1] * (szx[iz * nxt + ix] - szx[(iz - 1) * nxt + ix]);

            szx_x = dzi * hc[1] * (szx[iz * nxt + ix] - szx[iz * nxt + ix - 1]);
            szz_z = dzi * hc[1] * (szz[(iz + 1) * nxt + ix] - szz[iz * nxt + ix]);
            // ---------------------------------------------------
            // CPML layers for particle velocity kernel
            // ---------------------------------------------------

            if (npml > 0) {

                if (ix >= fpad && ix < ppad) { // left CPML
                  // Mapping the static CPML and memory variables to
                    px = ix - fpad; // the memory array index

                    mem_sxx_x[iz * 2 * (npml + 1) + px] = b[px] * mem_sxx_x[iz * 2 * (npml + 1) + px] + a[px] * sxx_x;
                    mem_szx_x[iz * 2 * (npml + 1) + px] = b_half[px] * mem_szx_x[iz * 2 * (npml + 1) + px] + a_half[px] * szx_x;

                    sxx_x = sxx_x / K[px] + mem_sxx_x[iz * 2 * (npml + 1) + px];
                    szx_x = szx_x / K_half[px] + mem_szx_x[iz * 2 * (npml + 1) + px];

                } // cpml left

                if (ix >= (nxt - ppad - 1) && ix < (nxt - fpad)) { // right CPML
                  // Mapping the static CPML and memory variables to
                    px = ix - pnx; // The PML factors index

                    mem_sxx_x[iz * 2 * (npml + 1) + px] = b[px] * mem_sxx_x[iz * 2 * (npml + 1) + px] + a[px] * sxx_x;
                    mem_szx_x[iz * 2 * (npml + 1) + px] = b_half[px] * mem_szx_x[iz * 2 * (npml + 1) + px] + a_half[px] * szx_x;

                    sxx_x = sxx_x / K[px] + mem_sxx_x[iz * 2 * (npml + 1) + px];
                    szx_x = szx_x / K_half[px] + mem_szx_x[iz * 2 * (npml + 1) + px];

                } // cpml right


                if (iz >= fpad && iz < ppad && !fsurf) { // top CPML
                  // Mapping the static CPML and memory variables to
                    pz = iz - fpad; // the memory array index

                    mem_szz_z[pz * nxt + ix] = b[pz] * mem_szz_z[pz * nxt + ix] + a[pz] * szz_z;
                    mem_szx_z[pz * nxt + ix] = b_half[pz] * mem_szx_z[pz * nxt + ix] + a_half[pz] * szx_z;

                    szz_z = szz_z / K[pz] + mem_szz_z[pz * nxt + ix];
                    szx_z = szx_z / K_half[pz] + mem_szx_z[pz * nxt + ix];

                } // cpml top

                if (iz >= (nzt - ppad - 1) && iz < nzt - fpad) { // bottom CPML
                  // Mapping the static CPML and memory variables to
                    pz = iz - pnz; // The PML factors index
                    mem_szz_z[pz * nxt + ix] = b[pz] * mem_szz_z[pz * nxt + ix] + a[pz] * szz_z;
                    mem_szx_z[pz * nxt + ix] = b_half[pz] * mem_szx_z[pz * nxt + ix] + a_half[pz] * szx_z;

                    szz_z = szz_z / K[pz] + mem_szz_z[pz * nxt + ix];
                    szx_z = szx_z / K_half[pz] + mem_szx_z[pz * nxt + ix];

                } // cpml bottom
                __syncthreads();
            } // npml>0

            // update particle velocities
            vx[iz * nxt + ix] += dt * rho_xp[iz * (nxt - 1) + ix] * (sxx_x + szx_z);
            vz[iz * nxt + ix] += dt * rho_zp[iz * (nxt - 1) + ix] * (szx_x + szz_z);


        }
        else { return; }

        //  break;

    default:
        //  std::cout << "FDORDER = " << fdorder << npml << std::endl;
        //  std::cout << "FD order mismatch. <SIMULATION ABORTED>" << std::endl;
          //exit(0);
       // printf("FD order mismatch. <SIMULATION ABORTED>\n");
    } // end of switch


}



__global__ void kernel_Thri(int nx1, int nx2, int fpad, int nxt, real_sim* szx, real_sim* szz) {
    int sz = blockIdx.x * blockDim.x + threadIdx.x;
    int ix = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (ix >= nx1 && ix < nx2 && sz >= 1 && sz <= fpad) {
       // printf("Hello Executed below thri \n");
        // mirroring 
        szx[(fpad - sz) * nxt + ix] = -szx[(fpad + sz) * nxt + ix];
        szz[(fpad - sz) * nxt + ix] = -szz[(fpad + sz) * nxt + ix];

    }

}









void forward_kernel_PSV_GPU(int ishot, // shot number
                        // Time and space grid arguments
    int nt, int nzt, int nxt, int fpad, int ppad,
    real_sim dt, real_sim dx, real_sim dz, int snap_interval, bool fsurf,
    // computationsl arguments
    real_sim* hc, int fdorder,
    // Wave arguments
    real_sim** vx, real_sim** vz,
    real_sim** sxx, real_sim** szx, real_sim** szz,
    // Medium arguments
    real_sim** lam, real_sim** mu,
    real_sim** mu_zx, real_sim** rho_zp, real_sim** rho_xp,
    //PML arguments
    int npml, real_sim* a, real_sim* b, real_sim* K,
    real_sim* a_half, real_sim* b_half, real_sim* K_half,
    // PML memory arrays
    real_sim** mem_vx_x, real_sim** mem_vx_z, real_sim** mem_vz_x, real_sim** mem_vz_z,
    real_sim** mem_sxx_x, real_sim** mem_szx_x, real_sim** mem_szz_z, real_sim** mem_szx_z,
    // Source arguments
    int nsrc, ivec src_x, ivec src_z, ivec src_comp,
    real_sim** src_signal, ivec source_to_fire_shot,
    // Receiver arguments
    int nrec, ivec rec_x, ivec rec_z, ivec rec_comp,
    real_sim** rec_signal, ivec receiver_to_record_shot,//shot_to_fire
    // FWI arguments
    bool fwinv, int fwi_dt, int fwi_dx, int fwi_dz,
    int fwi_x1, int fwi_x2, int fwi_z1, int fwi_z2,
    real_sim*** fwi_vx, real_sim*** fwi_vz, real_sim*** fwi_sxx,
    real_sim*** fwi_szx, real_sim*** fwi_szz,
    //*****************CPU PARAMS***************
    real_sim* d_a, real_sim* d_b, real_sim* d_K, real_sim* d_a_half, real_sim* d_b_half, real_sim* d_K_half,
    //
    real_sim* d_vx, real_sim* d_vz, real_sim* d_sxx, real_sim* d_szx, real_sim* d_szz,
    //
    real_sim* d_fwi_vx,
    real_sim* d_fwi_vz,
    real_sim* d_fwi_sxx,
    real_sim* d_fwi_szx,
    real_sim* d_fwi_szz,
    //
    real_sim* d_mem_vx_x, real_sim* d_mem_vx_z, real_sim* d_mem_vz_x, real_sim* d_mem_vz_z,
    real_sim* d_mem_sxx_x, real_sim* d_mem_szx_x, real_sim* d_mem_szz_z, real_sim* d_mem_szx_z,
    //
    real_sim* d_lam, real_sim* d_mu,
    real_sim* d_mu_zx, real_sim* d_rho_zp, real_sim* d_rho_xp

) {
    // std::cout << "Reached Here1" << "\n";
     //const bool fwi = 1;
     // int nt = number of timesteps
     // int nz1, nz2, nx1, nx2 = start and end grids along z and x directions
     // int dt, dx, dz = grid spacing in time and space
     // int* hc = holberg coefficients
     // real_sim **&vx, **&vz, **&sxx, **&szx, **&szz, // wave parameters (particle velocity and stresses)
     // real_sim **&lam, **&mu, **&mu_zx, **&rho_zp, **&rho_xp // medium parameters (lam�'s parameters')

     // real_sim *a, *b, *K;// CPML parameters
     // real_sim *a_half, *b_half, *K_half // CPML interpolated parameters
     // real_sim ** mem_vx_x, ** mem_vx_z, ** mem_vz_x, ** mem_vz_z; // PML velocity derivative memory
     // real_sim **&mem_sxx_x, **&mem_szx_x, **&mem_szz_z, real_sim **&mem_szx_z // PML stress derivative memory
     // bool fsurf :: free surface on the top


     // Source arguments
     // int nsrc = number of sources
     // int **src_loc = grid location of source + source parameter type for eg exploxive, vz only etc
     // real_sim ** src_signal = signal values for the sources

    real_sim sxx_x, szx_x, szx_z, szz_z; // spatial stress derivatives
    real_sim vx_x, vx_z, vz_x, vz_z; // spatial velocity derivatives
    int nz1, nz2, nx1, nx2; // The computational grid boundaries
    int px, pz; // index for PML arrys
    int isnap; // to take snapshots for data storage

    int tf, zf, xf; // Index parameters for fwi data storage

    std::ofstream outFile; // file to print vz arrays


    // Initial calculation of indices
    //---------------------------------------------
    nz1 = fpad; nz2 = nzt - fpad; nx1 = fpad; nx2 = nxt - fpad; // index variables

    // index to map PML at the right or positive end
    int pnx, pnz;
    pnx = nxt - 2 * ppad + fpad - 1; // nx + ppad + npml + 1 and nx = nxt - 2*ppad

    if (fsurf) {
        pnz = nzt - 2 * ppad - 1; // nz + ppad + npml + 1 and nz = nzt - ppad - fpad
    }
    else {
        pnz = nzt - 2 * ppad + fpad - 1; // nz + ppad + npml + 1 and nz = nzt - 2*ppad
    }

    real_sim dxi = 1.0 / dx; real_sim dzi = 1.0 / dz; // inverse of dx and dz

    // -----------------------------------------------------------

    // Reset kernels
    // -----------------------------------------------------

    real_sim size = nzt * nxt;
    gpuErrchk(hipMemset(d_vz, 0, size * sizeof(real_sim)));
    gpuErrchk(hipMemset(d_vx, 0, size * sizeof(real_sim)));

    gpuErrchk(hipMemset(d_sxx, 0, size * sizeof(real_sim)));
    gpuErrchk(hipMemset(d_szx, 0, size * sizeof(real_sim)));
    gpuErrchk(hipMemset(d_szz, 0, size * sizeof(real_sim)));


    
    if (fwinv) {

        const int nft = 1 + (nt - 1) / fwi_dt;
        const int nfz = 1 + (fwi_z2 - fwi_z1) / fwi_dz;
        const int nfx = 1 + (fwi_x2 - fwi_x1) / fwi_dx;
        size = nft * nfz * nfx;
 
        gpuErrchk(hipMemset(d_fwi_vx, 0, size * sizeof(real_sim)));
        gpuErrchk(hipMemset(d_fwi_vz, 0, size * sizeof(real_sim)));
        gpuErrchk(hipMemset(d_fwi_sxx, 0, size * sizeof(real_sim)));
        gpuErrchk(hipMemset(d_fwi_szx, 0, size * sizeof(real_sim)));
        gpuErrchk(hipMemset(d_fwi_szz, 0, size * sizeof(real_sim)));



    }

    // Start of time loop
    isnap = 0;
    //**********************************Memory Copy**************************************
    clock_t start = 0, end = 0;
    double start1 = 0;
    double cpu_time_used;

    size = nzt * nxt;

    int box1 = 16, box2 = 16;
    dim3 threadsPerBlock(box1, box2);
    dim3 blocksPerGrid((nz2-nz1 + box1 - 1) / box1, (nx2-nx1 + box2 - 1) / box2);

   box1 = 16, box2 = 16;
    dim3 threadsPerBlock0(box1, box2);
    dim3 blocksPerGrid0((fwi_z2- fwi_z1 + box1 - 1) / box1, (fwi_x2- fwi_x1 + box2 - 1) / box2);

 
    const int nft = 1 + (nt - 1) / fwi_dt;
    const int nfz = 1 + (fwi_z2 - fwi_z1) / fwi_dz;
    const int nfx = 1 + (fwi_x2 - fwi_x1) / fwi_dx;

    gpuErrchk(hipPeekAtLastError());
    //**************************************************************************************************


    for (int it = 0; it < nt; it++) {
        start = clock();

        // ---------------------------------------------------------
        // Storing velocity  & stress tensors for gradient calculations for full waveform inversion
        if (fwinv && !(it % fwi_dt)) {
            tf = it / fwi_dt; // t index for fwi gradient storage
           // std::cout<<"fwi time: " << it << ", forward simulation  "<<fwi_dt << std::endl;

            kernel_zero << < blocksPerGrid0, threadsPerBlock >> > (tf, nt, fwi_z1, fwi_z2, fwi_x1, fwi_x2,
                fwi_dz, fwi_dx, nft, nzt, nxt,nfx, d_fwi_sxx, d_fwi_szx, d_fwi_szz, d_fwi_vx,
                d_fwi_vz, d_sxx, d_szx, d_szz, d_vx, d_vz);
            gpuErrchk(hipPeekAtLastError());
        }


        //(int tf, int nt, int fwi_z1, int fwi_z2, int fwi_x1, int fwi_x2, int fwi_dz, int fwi_dx, int nft, int nzt, int nxt,nfx real_sim* fwi_sxx, real_sim* fwi_szx, real_sim* fwi_szz, real_sim* fwi_vx, real_sim*
       // fwi_vz, real_sim* sxx, real_sim* szx, real_sim* szz, real_sim* vx, real_sim* vz)
        gpuErrchk(hipDeviceSynchronize());
        //******************************************GPU****************************************


       gpuErrchk(hipMemcpy(d_vz, vz[0], size * sizeof(real_sim), hipMemcpyHostToDevice));

        // Calculate spatial velocity derivatives
        kernel_one << < blocksPerGrid, threadsPerBlock >> > (ishot, nt, nzt, nxt, fpad, ppad, dt, dx, dz,
            fdorder, d_vx, d_vz, d_sxx,
            d_szx, d_szz, d_lam, d_mu,
            d_mu_zx, d_rho_zp, d_rho_xp, npml,
            d_a, d_b, d_K, d_a_half, d_b_half, d_K_half,
            d_mem_vx_x, d_mem_vx_z, d_mem_vz_x, d_mem_vz_z,
            d_mem_sxx_x, d_mem_szx_x, d_mem_szz_z, d_mem_szx_z, fsurf);
      //  gpuErrchk(hipDeviceSynchronize());
        gpuErrchk(hipPeekAtLastError());

        // compute spatial stress derivatives
        kernel_two << < blocksPerGrid, threadsPerBlock >> > (ishot, nt, nzt, nxt, fpad, ppad, dt, dx, dz,
            fdorder, d_vx, d_vz, d_sxx,
            d_szx, d_szz, d_lam, d_mu,
            d_mu_zx, d_rho_zp, d_rho_xp, npml,
            d_a, d_b, d_K, d_a_half, d_b_half, d_K_half,
            d_mem_vx_x, d_mem_vx_z, d_mem_vz_x, d_mem_vz_z,
            d_mem_sxx_x, d_mem_szx_x, d_mem_szz_z, d_mem_szx_z, fsurf);
        gpuErrchk(hipDeviceSynchronize());
        gpuErrchk(hipPeekAtLastError());

        gpuErrchk(hipMemcpy(vz[0], d_vz, size * sizeof(real_sim), hipMemcpyDeviceToHost));

        if (fsurf) { // Mirroring stresses for free surface condition

            kernel_Thri << < blocksPerGrid, threadsPerBlock >> > (nx1, nx2, fpad, nxt, d_szx, d_szz);

        }

        gpuErrchk(hipDeviceSynchronize());
        gpuErrchk(hipPeekAtLastError());
        //****************************************************************************************************************




                // Adding Velocity update related sources

                //--------------------------
        for (int is = 0; is < nsrc; is++) {

            if (source_to_fire_shot[is] == ishot) {
                switch (src_comp[is]) {// defines the signal type
                case(2): // vz component only
                    vz[src_z[is]][src_x[is]] += src_signal[is][it];
                }
            }
        }

 

        // Recording the signals to the receivers
        for (int ir = 0; ir <= 0 /*nrec*/; ir++) {

            if (receiver_to_record_shot[ir] == ishot) {

                switch (rec_comp[ir]) {// defines the signal type
                case(2): // vz component only
                    rec_signal[ir][it] = vz[rec_z[ir]][rec_x[ir]];
                }

            }
        }


        // Printing out AASCII data for snap intervals
      /*  if (!(it % snap_interval || it == 0)) {
            std::cout << "Time step " << it << " of " << nt << " in forward kernel." << std::endl;
              outFile.open("./io/snap_data/vz_snap" + std::to_string(isnap) + ".csv");

              for (int j = 0; j < nzt; j++) {
                  for (int i = 0; i < nxt; i++) {
                      outFile << vz[j][i] << ", ";
                  }
                  outFile << std::endl;
              }
              outFile.close();
            isnap++;
        }*/
  
    } // end of time loop

    


}



// ****************               CPU    ****************************
//forward_kernel_PSV.cpp


void forward_kernel_PSV(int ishot, // shot number
                        // Time and space grid arguments
    int nt, int nzt, int nxt, int fpad, int ppad,
    real_sim dt, real_sim dx, real_sim dz, int snap_interval, bool fsurf,
    // computationsl arguments
    real_sim* hc, int fdorder,
    // Wave arguments
    real_sim** vx, real_sim** vz,
    real_sim** sxx, real_sim** szx, real_sim** szz,
    // Medium arguments
    real_sim** lam, real_sim** mu,
    real_sim** mu_zx, real_sim** rho_zp, real_sim** rho_xp,
    //PML arguments
    int npml, real_sim* a, real_sim* b, real_sim* K,
    real_sim* a_half, real_sim* b_half, real_sim* K_half,
    // PML memory arrays
    real_sim** mem_vx_x, real_sim** mem_vx_z, real_sim** mem_vz_x, real_sim** mem_vz_z,
    real_sim** mem_sxx_x, real_sim** mem_szx_x, real_sim** mem_szz_z, real_sim** mem_szx_z,
    // Source arguments
    int nsrc, ivec src_x, ivec src_z, ivec src_comp,
    real_sim** src_signal, ivec source_to_fire_shot,
    // Receiver arguments
    int nrec, ivec rec_x, ivec rec_z, ivec rec_comp,
    real_sim** rec_signal, ivec receiver_to_record_shot,//shot_to_fire
    // FWI arguments
    bool fwinv, int fwi_dt, int fwi_dx, int fwi_dz,
    int fwi_x1, int fwi_x2, int fwi_z1, int fwi_z2,
    real_sim*** fwi_vx, real_sim*** fwi_vz, real_sim*** fwi_sxx,
    real_sim*** fwi_szx, real_sim*** fwi_szz) {
    


     // Source arguments
     // int nsrc = number of sources
     // int **src_loc = grid location of source + source parameter type for eg exploxive, vz only etc
     // real_sim ** src_signal = signal values for the sources
    double start1 = clock();
    real_sim sxx_x, szx_x, szx_z, szz_z; // spatial stress derivatives
    real_sim vx_x, vx_z, vz_x, vz_z; // spatial velocity derivatives
    int nz1, nz2, nx1, nx2; // The computational grid boundaries
    int px, pz; // index for PML arrys
    int isnap; // to take snapshots for data storage

    int tf, zf, xf; // Index parameters for fwi data storage

    std::ofstream outFile; // file to print vz arrays
    clock_t start = 0, end = 0;
    double cpu_time_used;

    // Initial calculation of indices
    //---------------------------------------------
    nz1 = fpad; nz2 = nzt - fpad; nx1 = fpad; nx2 = nxt - fpad; // index variables

    // index to map PML at the right or positive end
    int pnx, pnz;
    pnx = nxt - 2 * ppad + fpad - 1; // nx + ppad + npml + 1 and nx = nxt - 2*ppad

    if (fsurf) {
        pnz = nzt - 2 * ppad - 1; // nz + ppad + npml + 1 and nz = nzt - ppad - fpad
    }
    else {
        pnz = nzt - 2 * ppad + fpad - 1; // nz + ppad + npml + 1 and nz = nzt - 2*ppad
    }

    real_sim dxi = 1.0 / dx; real_sim dzi = 1.0 / dz; // inverse of dx and dz

    // -----------------------------------------------------------

    // Reset kernels
    // -----------------------------------------------------
   //****************************************************************************************


    //**************************************************************************************************


    for (int iz = 0; iz < nzt; iz++) {
        for (int ix = 0; ix < nxt; ix++) {
            // Wave velocity and stress tensor arrays
            vx[iz][ix] = 0.0;
            vz[iz][ix] = 0.0;
            sxx[iz][ix] = 0.0;
            szx[iz][ix] = 0.0;
            szz[iz][ix] = 0.0;

        }
    }
    // std::cout << "Reached Here1.5" << "\n";
    if (fwinv) {
        const int nft = 1 + (nt - 1) / fwi_dt;
        const int nfz = 1 + (fwi_z2 - fwi_z1) / fwi_dz;
        const int nfx = 1 + (fwi_x2 - fwi_x1) / fwi_dx;

        for (int it = 0; it < nft; it++) {
            for (int iz = 0; iz < nfz; iz++) {
                for (int ix = 0; ix < nfx; ix++) {

                    // FWI storage arrays
                    fwi_vx[it][iz][ix] = 0.0;
                    fwi_vz[it][iz][ix] = 0.0;
                    fwi_sxx[it][iz][ix] = 0.0;
                    fwi_szx[it][iz][ix] = 0.0;
                    fwi_szz[it][iz][ix] = 0.0;

                }
            }
        }

    }

    // Start of time loop
    isnap = 0;

    for (int it = 0; it < nt; it++) {

        // ---------------------------------------------------------
        // Storing velocity  & stress tensors for gradient calculations for full waveform inversion
        if (fwinv && !(it % fwi_dt)) {
            tf = it / fwi_dt; // t index for fwi gradient storage

           std::cout<<"fwi time: " << it << ", forward simulation" << std::endl;
            for (int iz = fwi_z1; iz < fwi_z2; iz += fwi_dz) { // storing only a portion and with grid inteval
                zf = (iz - fwi_z1) / fwi_dz; // z index for fwi gradient storage

                for (int ix = fwi_x1; ix < fwi_x2; ix += fwi_dx) {
                    xf = (ix - fwi_x1) / fwi_dx; // x index for fwi gradient storage

                    fwi_sxx[tf][zf][xf] = sxx[iz][ix];
                    fwi_szx[tf][zf][xf] = szx[iz][ix];
                    fwi_szz[tf][zf][xf] = szz[iz][ix];
                    fwi_vx[tf][zf][xf] = vx[iz][ix];
                    fwi_vz[tf][zf][xf] = vz[iz][ix];

                }
            }
        }

        // --------------------------------------------------------
      //  std::cout << "Reached Here2" << "\n";
        start = clock();
        // Time integration of dynamic stress fields
        switch (fdorder) {
        case(2):
            // updating stress tensors
            for (int iz = nz1; iz < nz2; iz++) {
                //std::cout << std::endl << "PML indices: " << std::endl;
                for (int ix = nx1; ix < nx2; ix++) {
                    //std::cout << ix << ", " << iz << std::endl;

                    // Calculate spatial velocity derivatives
                    vx_x = dxi * hc[1] * (vx[iz][ix] - vx[iz][ix - 1]);
                    vz_x = dxi * hc[1] * (vz[iz][ix + 1] - vz[iz][ix]);
                    vx_z = dzi * hc[1] * (vx[iz + 1][ix] - vx[iz][ix]);
                    vz_z = dzi * hc[1] * (vz[iz][ix] - vz[iz - 1][ix]);

                    // ---------------------------------------------------
                    // CPML layers for stress tensor kernel
                    // ---------------------------------------------------

                    if (npml > 0) {

                        if (ix >= fpad && ix <= ppad) { // left CPML
                          // Mapping the static CPML and memory variables to
                            px = ix - fpad; // the memory array index
                            //std::cout << std::endl << "Fault1 " << ix << std::endl;

                            mem_vx_x[iz][px] = b[px] * mem_vx_x[iz][px] + a[px] * vx_x;
                            mem_vz_x[iz][px] = b_half[px] * mem_vz_x[iz][px] + a_half[px] * vz_x;

                            vx_x = vx_x / K[px] + mem_vx_x[iz][px];
                            vz_x = vz_x / K_half[px] + mem_vz_x[iz][px];

                        } // cpml left

                        if (ix >= (nxt - ppad - 1) && ix < nxt - fpad) { // right CPML
                          // Mapping the static CPML and memory variables to

                            px = ix - pnx; // The PML factors index

                            mem_vx_x[iz][px] = b[px] * mem_vx_x[iz][px] + a[px] * vx_x;
                            mem_vz_x[iz][px] = b_half[px] * mem_vz_x[iz][px] + a_half[px] * vz_x;

                            vx_x = vx_x / K[px] + mem_vx_x[iz][px];
                            vz_x = vz_x / K_half[px] + mem_vz_x[iz][px];


                        } // cpml right


                        if (iz >= fpad && iz <= ppad && !fsurf) { // top CPML
                          // Mapping the static CPML and memory variables to
                            pz = iz - fpad; // the memory array index


                            mem_vz_z[pz][ix] = b[pz] * mem_vz_z[pz][ix] + a[pz] * vz_z;
                            mem_vx_z[pz][ix] = b_half[pz] * mem_vx_z[pz][ix] + a_half[pz] * vx_z;

                            vz_z = vz_z / K[pz] + mem_vz_z[pz][ix];
                            vx_z = vx_z / K_half[pz] + mem_vx_z[pz][ix];

                            //std::cout << pz<< ", ";

                        } // cpml top

                        if (iz >= (nzt - ppad - 1) && iz < nzt - fpad) { // bottom CPML
                          // Mapping the static CPML and memory variables to
                            pz = iz - pnz; // The PML factors index

                            mem_vz_z[pz][ix] = b[pz] * mem_vz_z[pz][ix] + a[pz] * vz_z;
                            mem_vx_z[pz][ix] = b_half[pz] * mem_vx_z[pz][ix] + a_half[pz] * vx_z;

                            vz_z = vz_z / K[pz] + mem_vz_z[pz][ix];
                            vx_z = vx_z / K_half[pz] + mem_vx_z[pz][ix];

                            //std::cout << pz<< ", ";

                        } // cpml bottom

                    } // npml>0
                    // --------------------------------------------------------------------------
                    // --------------------------------------------------------------------------

                    // updating stresses
                    szx[iz][ix] += dt * mu_zx[iz][ix] * (vz_x + vx_z);
                    sxx[iz][ix] += dt * (lam[iz][ix] * (vx_x + vz_z) + (2.0 * mu[iz][ix] * vx_x));
                    szz[iz][ix] += dt * (lam[iz][ix] * (vx_x + vz_z) + (2.0 * mu[iz][ix] * vz_z));

                    // -----------------------------------------
                    // ----------------------------------------- 
                    // Override stress for free surface implementation

                    if (fsurf && iz == fpad) {
                        // Free surface at z = 0 or nz = fpad

                        // Denise manual  page 13
                        szz[fpad][ix] = 0.0;
                        szx[fpad][ix] = 0.0;
                        sxx[fpad][ix] = 4.0 * dt * vx_x * (lam[fpad][ix] * mu[fpad][ix] + mu[fpad][ix] * mu[fpad][ix])
                            / (lam[fpad][ix] + 2.0 * mu[fpad][ix]);

                        //}
                    }


                }
            }


            // STRESS MIRRORING TECHNIQUE FOR FREE SURFACE CONDITION
            if (fsurf) { // Mirroring stresses for free surface condition
                for (int ix = nx1; ix < nx2; ix++) {
                    for (int sz = 1; sz <= fpad; sz++) { // mirroring 
                        szx[fpad - sz][ix] = -szx[fpad + sz][ix];
                        szz[fpad - sz][ix] = -szz[fpad + sz][ix];
                    }
                }
            }

            // std::cout << "Reached Here3" << "\n";

             // updating velocity tensors
            for (int iz = nz1; iz < nz2; iz++) {
                for (int ix = nx1; ix < nx2; ix++) {

                    // compute spatial stress derivatives
                    sxx_x = dxi * hc[1] * (sxx[iz][ix + 1] - sxx[iz][ix]);
                    szx_x = dxi * hc[1] * (szx[iz][ix] - szx[iz][ix - 1]);
                    szx_z = dzi * hc[1] * (szx[iz][ix] - szx[iz - 1][ix]);
                    szz_z = dzi * hc[1] * (szz[iz + 1][ix] - szz[iz][ix]);


                    // ---------------------------------------------------
                    // CPML layers for particle velocity kernel
                    // ---------------------------------------------------

                    if (npml > 0) {

                        if (ix >= fpad && ix < ppad) { // left CPML
                          // Mapping the static CPML and memory variables to
                            px = ix - fpad; // the memory array index

                            mem_sxx_x[iz][px] = b[px] * mem_sxx_x[iz][px] + a[px] * sxx_x;
                            mem_szx_x[iz][px] = b_half[px] * mem_szx_x[iz][px] + a_half[px] * szx_x;

                            sxx_x = sxx_x / K[px] + mem_sxx_x[iz][px];
                            szx_x = szx_x / K_half[px] + mem_szx_x[iz][px];

                        } // cpml left

                        if (ix >= (nxt - ppad - 1) && ix < nxt - fpad) { // right CPML
                          // Mapping the static CPML and memory variables to
                            px = ix - pnx; // The PML factors index

                            mem_sxx_x[iz][px] = b[px] * mem_sxx_x[iz][px] + a[px] * sxx_x;
                            mem_szx_x[iz][px] = b_half[px] * mem_szx_x[iz][px] + a_half[px] * szx_x;

                            sxx_x = sxx_x / K[px] + mem_sxx_x[iz][px];
                            szx_x = szx_x / K_half[px] + mem_szx_x[iz][px];

                        } // cpml right

                        if (iz >= fpad && iz < ppad && !fsurf) { // top CPML
                          // Mapping the static CPML and memory variables to
                            pz = iz - fpad; // the memory array index

                            mem_szz_z[pz][ix] = b[pz] * mem_szz_z[pz][ix] + a[pz] * szz_z;
                            mem_szx_z[pz][ix] = b_half[pz] * mem_szx_z[pz][ix] + a_half[pz] * szx_z;

                            szz_z = szz_z / K[pz] + mem_szz_z[pz][ix];
                            szx_z = szx_z / K_half[pz] + mem_szx_z[pz][ix];

                        } // cpml top

                        if (iz >= (nzt - ppad - 1) && iz < nzt - fpad) { // bottom CPML
                          // Mapping the static CPML and memory variables to
                            pz = iz - pnz; // The PML factors index

                            mem_szz_z[pz][ix] = b[pz] * mem_szz_z[pz][ix] + a[pz] * szz_z;
                            mem_szx_z[pz][ix] = b_half[pz] * mem_szx_z[pz][ix] + a_half[pz] * szx_z;

                            szz_z = szz_z / K[pz] + mem_szz_z[pz][ix];
                            szx_z = szx_z / K_half[pz] + mem_szx_z[pz][ix];

                        } // cpml bottom

                    } // npml>0
                    // --------------------------------------------------------------------------
                    // --------------------------------------------------------------------------

                    // update particle velocities
                    vx[iz][ix] += dt * rho_xp[iz][ix] * (sxx_x + szx_z);
                    vz[iz][ix] += dt * rho_zp[iz][ix] * (szx_x + szz_z);

                }
            }

            break;

        default:
            std::cout << "FDORDER = " << fdorder << npml << std::endl;
            std::cout << "FD order mismatch. <SIMULATION ABORTED>" << std::endl;
            exit(0);
        } // end of switch


        end = clock();
        cpu_time_used = ((double)(end - start)) / CLOCKS_PER_SEC;

        //  printf("\nExecution time Gpu = %f\n", cpu_time_used);


        start = clock();

        // Adding Velocity update related sources
     //   std::cout << "Reached Here4" << "\n";
        //--------------------------
        for (int is = 0; is < nsrc; is++) {

            if (source_to_fire_shot[is] == ishot) {
                switch (src_comp[is]) {// defines the signal type
                case(2): // vz component only
                    vz[src_z[is]][src_x[is]] += src_signal[is][it];
                }
            }
        }

        // ------------------------------------

        // Recording the signals to the receivers
        for (int ir = 0; ir <=0 /*nrec*/; ir++) {
           
            if (receiver_to_record_shot[ir] == ishot) {
               
                switch (rec_comp[ir]) {// defines the signal type
                case(2): // vz component only
                    rec_signal[ir][it] = vz[rec_z[ir]][rec_x[ir]];
                }

            }
        }


        // Printing out AASCII data for snap intervals
        if (!(it % snap_interval || it == 0)) {
            std::cout << "Time step " << it << " of " << nt << " in forward kernel." << std::endl;
            //outFile.open("./io/snap_data/vz_snap" + std::to_string(isnap) + ".csv");
          /*  for (int j = 0; j < nzt; j++) {
                for (int i = 0; i < nxt; i++) {
                    outFile << vz[j][i] << ", ";
                }
                outFile << std::endl;
            }
            outFile.close();*/
            isnap++;
        }
        /*end = clock();
        cpu_time_used = ((double)(end - start)) / CLOCKS_PER_SEC;*/

        //  printf("\nExecution time after Gpu = %f\n", cpu_time_used);



          // std::cout << "Reached Here6" << "\n";
    } // end of time loop
    end = clock();
    cpu_time_used = ((double)(end - start1)) / CLOCKS_PER_SEC;

   // printf("\nKERNEL TIME = %f\n", cpu_time_used);


}