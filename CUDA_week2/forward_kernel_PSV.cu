#include "hip/hip_runtime.h"

/*
* Created by: Min Basnet
* 2020.April.16
* Kathmandu, Nepal
*/
#include <time.h>
#include <iostream>
#include <fstream>
#include <cmath>
#include <vector>

#include "globvar.cuh"
#include "util.cu"
#include "fd_cpml.cuh"
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}



__global__ void kernel_zero(int tf, int nzt, int fwi_z1, int fwi_z2, int fwi_x1, int fwi_x2, int fwi_dz, int fwi_dx, int nft, int nfz, int nfx, real_sim* fwi_sxx, real_sim* fwi_szx, real_sim* fwi_szz, real_sim* fwi_vx, real_sim*
    fwi_vz, real_sim* sxx, real_sim* szx, real_sim* szz, real_sim* vx, real_sim* vz) {

    int iz = blockIdx.x * blockDim.x + threadIdx.x;
    int ix = blockIdx.y * blockDim.y + threadIdx.y;

    if (iz >= fwi_z1 && iz < fwi_z2 && iz % fwi_dz == fwi_z1) { // storing only a portion and with grid inteval
        int zf = (iz - fwi_z1) / fwi_dz; // z index for fwi gradient storage

        if (ix >= fwi_x1 && ix < fwi_x2 && ix % fwi_dx == fwi_x1) {
            int xf = (ix - fwi_x1) / fwi_dx; // x index for fwi gradient storage
            int offset = tf * nft * nfz + zf * nfz + xf;
            fwi_sxx[offset] = sxx[iz * nzt + ix];

            fwi_szx[offset] = szx[iz * nzt + ix];
            fwi_szz[offset] = szz[iz * nzt + ix];

            fwi_vx[offset] = vx[iz * nzt + ix];
            fwi_vz[offset] = vz[iz * nzt + ix];

        }
    }
    else {
        return;
    }
}

__global__ void kernel_one(int ishot, int nt, int nzt, int nxt, int fpad, int ppad, real_sim dt, real_sim dx, real_sim dz,
    int fdorder, real_sim* vx, real_sim* vz, real_sim* sxx,
    real_sim* szx, real_sim* szz, real_sim* lam, real_sim* mu,
    real_sim* mu_zx, real_sim* rho_zp, real_sim* rho_xp, int npml,
    real_sim* a, real_sim* b, real_sim* K, real_sim* a_half, real_sim* b_half, real_sim* K_half,
    real_sim* mem_vx_x, real_sim* mem_vx_z, real_sim* mem_vz_x, real_sim* mem_vz_z,
    real_sim* mem_sxx_x, real_sim* mem_szx_x, real_sim* mem_szz_z, real_sim* mem_szx_z, bool fsurf) {

    //**********************************************************************************
    real_sim sxx_x, szx_x, szx_z, szz_z; // spatial stress derivatives
    real_sim vx_x, vx_z, vz_x, vz_z; // spatial velocity derivatives
    int nz1, nz2, nx1, nx2; // The computational grid boundaries
    int px, pz; // index for PML arrys
    int isnap; // to take snapshots for data storage

    int tf, zf, xf; // Index parameters for fwi data storage

    real_sim hc[2] = { 1.0, 1.0 };

    // Initial calculation of indices
    //---------------------------------------------
    nz1 = fpad; nz2 = nzt - fpad; nx1 = fpad; nx2 = nxt - fpad; // index variables

    // index to map PML at the right or positive end
    int pnx, pnz;
    pnx = nxt - 2 * ppad + fpad - 1; // nx + ppad + npml + 1 and nx = nxt - 2*ppad

    if (fsurf) {
        pnz = nzt - 2 * ppad - 1; // nz + ppad + npml + 1 and nz = nzt - ppad - fpad
    }
    else {
        pnz = nzt - 2 * ppad + fpad - 1; // nz + ppad + npml + 1 and nz = nzt - 2*ppad
    }

    real_sim dxi = 1.0 / dx; real_sim dzi = 1.0 / dz; // inverse of dx and dz
//**********************************************************************************************************


    int iz = blockIdx.x * blockDim.x + threadIdx.x;
    int ix = blockIdx.y * blockDim.y + threadIdx.y;


    switch (fdorder) {
    case(2):
        if (ix < nx2 && ix >= nx1 && iz >= nz1 && iz < nz2)

        {

            // Calculate spatial velocity derivatives
            vx_x = dxi * hc[1] * (vx[iz * nzt + ix] - vx[iz * nzt + (ix - 1)]);
            vz_x = dxi * hc[1] * (vz[iz * nzt + (ix + 1)] - vz[iz * nzt + ix]);
            vx_z = dzi * hc[1] * (vx[(iz + 1) * nzt + ix] - vx[iz * nzt + ix]);
            vz_z = dzi * hc[1] * (vz[iz * nzt + ix] - vz[(iz - 1) * nzt + ix]);
            //************************************************************************************************

    // ---------------------------------------------------
    // CPML layers for stress tensor kernel
    // ---------------------------------------------------




        // ---------------------------------------------------
        // CPML layers for stress tensor kernel
         //---------------------------------------------------

            if (npml > 0) {

                if (ix >= fpad && ix <= ppad)
                { // left CPML
                  // Mapping the static CPML and memory variables to
                    px = ix - fpad; // the memory array index
                    //std::cout << std::endl << "Fault1 " << ix << std::endl;


                    mem_vx_x[iz * nzt + px] = b[px] * mem_vx_x[iz * nzt + px] + a[px] * vx_x;
                    mem_vz_x[iz * nzt + px] = b_half[px] * mem_vz_x[iz * nzt + px] + a_half[px] * vz_x;

                    vx_x = vx_x / K[px] + mem_vx_x[iz * nzt + px];
                    vz_x = vz_x / K_half[px] + mem_vz_x[iz * nzt + px];

                } // cpml left



                if (ix >= (nxt - ppad - 1) && ix < nxt - fpad) { // right CPML
                  // Mapping the static CPML and memory variables to

                    px = ix - pnx; // The PML factors index

                    mem_vx_x[iz * nzt + px] = b[px] * mem_vx_x[iz * nzt + px] + a[px] * vx_x;
                    mem_vz_x[iz * nzt + px] = b_half[px] * mem_vz_x[iz * nzt + px] + a_half[px] * vz_x;

                    vx_x = vx_x / K[px] + mem_vx_x[iz * nzt + px];
                    vz_x = vz_x / K_half[px] + mem_vz_x[iz * nzt + px];


                } // cpml right


                if (iz >= fpad && iz <= ppad && !fsurf) { // top CPML
                  // Mapping the static CPML and memory variables to
                    pz = iz - fpad; // the memory array index

                    mem_vz_z[pz * 2 * (npml + 1) + ix] = b[pz] * mem_vz_z[pz * 2 * (npml + 1) + ix] + a[pz] * vz_z;
                    mem_vx_z[pz * 2 * (npml + 1) + ix] = b_half[pz] * mem_vx_z[pz * 2 * (npml + 1) + ix] + a_half[pz] * vx_z;

                    vz_z = vz_z / K[pz] + mem_vz_z[pz * 2 * (npml + 1) + ix];
                    vx_z = vx_z / K_half[pz] + mem_vx_z[pz * 2 * (npml + 1) + ix];

                    //std::cout << pz<< ", ";

                } // cpml top



                if (iz >= (nzt - ppad - 1) && iz < nzt - fpad) { // bottom CPML
                  // Mapping the static CPML and memory variables to
                    pz = iz - pnz; // The PML factors index

                    mem_vz_z[pz * 2 * (npml + 1) + ix] = b[pz] * mem_vz_z[pz * 2 * (npml + 1) + ix] + a[pz] * vz_z;
                    mem_vx_z[pz * 2 * (npml + 1) + ix] = b_half[pz] * mem_vx_z[pz * 2 * (npml + 1) + ix] + a_half[pz] * vx_z;

                    vz_z = vz_z / K[pz] + mem_vz_z[pz * 2 * (npml + 1) + ix];
                    vx_z = vx_z / K_half[pz] + mem_vx_z[pz * 2 * (npml + 1) + ix];

                    //std::cout << pz<< ", ";

                } // cpml bottom

            }    // npml>0
            __syncthreads();
            
            ////// updating stresses

            szx[iz * nzt + ix] += dt * mu_zx[iz * (nzt - 1) + ix] * (vz_x + vx_z);
            sxx[iz * nzt + ix] += dt * (lam[iz * nzt + ix] * (vx_x + vz_z) + (2.0f * mu[iz * nzt + ix] * vx_x));
            szz[iz * nzt + ix] += dt * (lam[iz * nzt + ix] * (vx_x + vz_z) + (2.0f * mu[iz * nzt + ix] * vz_z));


   // Override stress for free surface implementation

            if (fsurf && iz == fpad) {
                // Free surface at z = 0 or nz = fpad

                // Denise manual  page 13
                szz[fpad * nzt + ix] = 0.0;
                szx[fpad * nzt + ix] = 0.0;
                sxx[fpad * nzt + ix] = 4.0 * dt * vx_x * (lam[fpad * nzt + ix] * mu[fpad * nzt + ix] + mu[fpad * nzt + ix] * mu[fpad * nzt + ix])
                    / (lam[fpad * nzt + ix] + 2.0 * mu[fpad * nzt + ix]);



            }


        }
        else {
            return;
        }

    default:
        //  std::cout << "FDORDER = " << fdorder << npml << std::endl;
        //  std::cout << "FD order mismatch. <SIMULATION ABORTED>" << std::endl;
          //exit(0);
       // printf("FD order mismatch. <SIMULATION ABORTED>\n");
    } // end of switch


}

__global__ void kernel_two(int ishot, int nt, int nzt, int nxt, int fpad, int ppad, real_sim dt, real_sim dx, real_sim dz,
    int fdorder, real_sim* vx, real_sim* vz, real_sim* sxx,
    real_sim* szx, real_sim* szz, real_sim* lam, real_sim* mu,
    real_sim* mu_zx, real_sim* rho_zp, real_sim* rho_xp, int npml,
    real_sim* a, real_sim* b, real_sim* K, real_sim* a_half, real_sim* b_half, real_sim* K_half,
    real_sim* mem_vx_x, real_sim* mem_vx_z, real_sim* mem_vz_x, real_sim* mem_vz_z,
    real_sim* mem_sxx_x, real_sim* mem_szx_x, real_sim* mem_szz_z, real_sim* mem_szx_z, bool fsurf) {


    real_sim sxx_x, szx_x, szx_z, szz_z; // spatial stress derivatives
    real_sim vx_x, vx_z, vz_x, vz_z; // spatial velocity derivatives
    int nz1, nz2, nx1, nx2; // The computational grid boundaries
    int px, pz; // index for PML arrys
    int isnap; // to take snapshots for data storage

    int tf, zf, xf; // Index parameters for fwi data storage


    real_sim hc[2] = { 1.0, 1.0 };

    // Initial calculation of indices
    //---------------------------------------------
    nz1 = fpad; nz2 = nzt - fpad; nx1 = fpad; nx2 = nxt - fpad; // index variables

    // index to map PML at the right or positive end
    int pnx, pnz;
    pnx = nxt - 2 * ppad + fpad - 1; // nx + ppad + npml + 1 and nx = nxt - 2*ppad

    if (fsurf) {
        pnz = nzt - 2 * ppad - 1; // nz + ppad + npml + 1 and nz = nzt - ppad - fpad
    }
    else {
        pnz = nzt - 2 * ppad + fpad - 1; // nz + ppad + npml + 1 and nz = nzt - 2*ppad
    }

    real_sim dxi = 1.0 / dx; real_sim dzi = 1.0 / dz; // inverse of dx and dz


    //*************************************************

    int iz = blockIdx.x * blockDim.x + threadIdx.x;
    int ix = blockIdx.y * blockDim.y + threadIdx.y;


    switch (fdorder) {
    case(2):
        if (ix < nx2 && ix >= nx1 && iz >= nz1 && iz < nz2)
        {
            // printf("abc");
            // compute spatial stress derivatives
            sxx_x = dxi * hc[1] * (sxx[iz * nzt + ix + 1] - sxx[iz * nzt + ix]);
            szx_z = dxi * hc[1] * (szx[iz * nzt + ix] - szx[(iz - 1) * nzt + ix]);

            szx_x = dzi * hc[1] * (szx[iz * nzt + ix] - szx[iz * nzt + ix - 1]);
            szz_z = dzi * hc[1] * (szz[(iz + 1) * nzt + ix] - szz[iz * nzt + ix]);


            // ---------------------------------------------------
            // CPML layers for particle velocity kernel
            // ---------------------------------------------------

            if (npml > 0) {

                if (ix >= fpad && ix < ppad) { // left CPML
                  // Mapping the static CPML and memory variables to
                    px = ix - fpad; // the memory array index

                    mem_sxx_x[iz * nzt + px] = b[px] * mem_sxx_x[iz * nzt + px] + a[px] * sxx_x;
                    mem_szx_x[iz * nzt + px] = b_half[px] * mem_szx_x[iz * nzt + px] + a_half[px] * szx_x;

                    sxx_x = sxx_x / K[px] + mem_sxx_x[iz * nzt + px];
                    szx_x = szx_x / K_half[px] + mem_szx_x[iz * nzt + px];

                } // cpml left

                if (ix >= (nxt - ppad - 1) && ix < (nxt - fpad)) { // right CPML
                  // Mapping the static CPML and memory variables to
                    px = ix - pnx; // The PML factors index

                    mem_sxx_x[iz * nzt + px] = b[px] * mem_sxx_x[iz * nzt + px] + a[px] * sxx_x;
                    mem_szx_x[iz * nzt + px] = b_half[px] * mem_szx_x[iz * nzt + px] + a_half[px] * szx_x;

                    sxx_x = sxx_x / K[px] + mem_sxx_x[iz * nzt + px];
                    szx_x = szx_x / K_half[px] + mem_szx_x[iz * nzt + px];

                } // cpml right


                if (iz >= fpad && iz < ppad && !fsurf) { // top CPML
                  // Mapping the static CPML and memory variables to
                    pz = iz - fpad; // the memory array index

                    mem_szz_z[pz * 2 * (npml + 1) + ix] = b[pz] * mem_szz_z[pz * 2 * (npml + 1) + ix] + a[pz] * szz_z;
                    mem_szx_z[pz * 2 * (npml + 1) + ix] = b_half[pz] * mem_szx_z[pz * 2 * (npml + 1) + ix] + a_half[pz] * szx_z;

                    szz_z = szz_z / K[pz] + mem_szz_z[pz * 2 * (npml + 1) + ix];
                    szx_z = szx_z / K_half[pz] + mem_szx_z[pz * 2 * (npml + 1) + ix];

                } // cpml top

                if (iz >= (nzt - ppad - 1) && iz < nzt - fpad) { // bottom CPML
                  // Mapping the static CPML and memory variables to
                    pz = iz - pnz; // The PML factors index

                    mem_szz_z[pz * 2 * (npml + 1) + ix] = b[pz] * mem_szz_z[pz * 2 * (npml + 1) + ix] + a[pz] * szz_z;
                    mem_szx_z[pz * 2 * (npml + 1) + ix] = b_half[pz] * mem_szx_z[pz * 2 * (npml + 1) + ix] + a_half[pz] * szx_z;

                    szz_z = szz_z / K[pz] + mem_szz_z[pz * 2 * (npml + 1) + ix];
                    szx_z = szx_z / K_half[pz] + mem_szx_z[pz * 2 * (npml + 1) + ix];

                } // cpml bottom
                __syncthreads();
            } // npml>0

            // update particle velocities
            vx[iz * nzt + ix] += dt * rho_xp[iz * (nzt - 1) + ix] * (sxx_x + szx_z);
            vz[iz * nzt + ix] += dt * rho_zp[iz * (nzt - 1) + ix] * (szx_x + szz_z);


        }
        else { return; }

        //  break;

    default:
        //  std::cout << "FDORDER = " << fdorder << npml << std::endl;
        //  std::cout << "FD order mismatch. <SIMULATION ABORTED>" << std::endl;
          //exit(0);
       // printf("FD order mismatch. <SIMULATION ABORTED>\n");
    } // end of switch


}



__global__ void kernel_Thri(int nx1, int nx2, int fpad, int nzt, real_sim* szx, real_sim* szz) {
    int iz = blockIdx.x * blockDim.x + threadIdx.x;
    int ix = blockIdx.y * blockDim.y + threadIdx.y;
    int sz = 1;
    if (ix >= nx1 && ix < nx2 && sz >= 1 && sz <= fpad) {
        // mirroring 
        szx[(fpad - sz) * nzt + ix] = -szx[(fpad + sz) * nzt + ix];
        szz[(fpad - sz) * nzt + ix] = -szz[(fpad + sz) * nzt + ix];

    }

}









void forward_kernel_PSV(int ishot, // shot number
                        // Time and space grid arguments
    int nt, int nzt, int nxt, int fpad, int ppad,
    real_sim dt, real_sim dx, real_sim dz, int snap_interval, bool fsurf,
    // computationsl arguments
    real_sim* hc, int fdorder,
    // Wave arguments
    real_sim** vx, real_sim** vz,
    real_sim** sxx, real_sim** szx, real_sim** szz,
    // Medium arguments
    real_sim** lam, real_sim** mu,
    real_sim** mu_zx, real_sim** rho_zp, real_sim** rho_xp,
    //PML arguments
    int npml, real_sim* a, real_sim* b, real_sim* K,
    real_sim* a_half, real_sim* b_half, real_sim* K_half,
    // PML memory arrays
    real_sim** mem_vx_x, real_sim** mem_vx_z, real_sim** mem_vz_x, real_sim** mem_vz_z,
    real_sim** mem_sxx_x, real_sim** mem_szx_x, real_sim** mem_szz_z, real_sim** mem_szx_z,
    // Source arguments
    int nsrc, ivec src_x, ivec src_z, ivec src_comp,
    real_sim** src_signal, ivec source_to_fire_shot,
    // Receiver arguments
    int nrec, ivec rec_x, ivec rec_z, ivec rec_comp,
    real_sim** rec_signal, ivec receiver_to_record_shot,//shot_to_fire
    // FWI arguments
    bool fwinv, int fwi_dt, int fwi_dx, int fwi_dz,
    int fwi_x1, int fwi_x2, int fwi_z1, int fwi_z2,
    real_sim*** fwi_vx, real_sim*** fwi_vz, real_sim*** fwi_sxx,
    real_sim*** fwi_szx, real_sim*** fwi_szz) {
    // std::cout << "Reached Here1" << "\n";
     //const bool fwi = 1;
     // int nt = number of timesteps
     // int nz1, nz2, nx1, nx2 = start and end grids along z and x directions
     // int dt, dx, dz = grid spacing in time and space
     // int* hc = holberg coefficients
     // real_sim **&vx, **&vz, **&sxx, **&szx, **&szz, // wave parameters (particle velocity and stresses)
     // real_sim **&lam, **&mu, **&mu_zx, **&rho_zp, **&rho_xp // medium parameters (lam�'s parameters')

     // real_sim *a, *b, *K;// CPML parameters
     // real_sim *a_half, *b_half, *K_half // CPML interpolated parameters
     // real_sim ** mem_vx_x, ** mem_vx_z, ** mem_vz_x, ** mem_vz_z; // PML velocity derivative memory
     // real_sim **&mem_sxx_x, **&mem_szx_x, **&mem_szz_z, real_sim **&mem_szx_z // PML stress derivative memory
     // bool fsurf :: free surface on the top


     // Source arguments
     // int nsrc = number of sources
     // int **src_loc = grid location of source + source parameter type for eg exploxive, vz only etc
     // real_sim ** src_signal = signal values for the sources

    real_sim sxx_x, szx_x, szx_z, szz_z; // spatial stress derivatives
    real_sim vx_x, vx_z, vz_x, vz_z; // spatial velocity derivatives
    int nz1, nz2, nx1, nx2; // The computational grid boundaries
    int px, pz; // index for PML arrys
    int isnap; // to take snapshots for data storage

    int tf, zf, xf; // Index parameters for fwi data storage

    std::ofstream outFile; // file to print vz arrays


    // Initial calculation of indices
    //---------------------------------------------
    nz1 = fpad; nz2 = nzt - fpad; nx1 = fpad; nx2 = nxt - fpad; // index variables

    // index to map PML at the right or positive end
    int pnx, pnz;
    pnx = nxt - 2 * ppad + fpad - 1; // nx + ppad + npml + 1 and nx = nxt - 2*ppad

    if (fsurf) {
        pnz = nzt - 2 * ppad - 1; // nz + ppad + npml + 1 and nz = nzt - ppad - fpad
    }
    else {
        pnz = nzt - 2 * ppad + fpad - 1; // nz + ppad + npml + 1 and nz = nzt - 2*ppad
    }

    real_sim dxi = 1.0 / dx; real_sim dzi = 1.0 / dz; // inverse of dx and dz

    // -----------------------------------------------------------

    // Reset kernels
    // -----------------------------------------------------

    for (int iz = 0; iz < nzt; iz++) {
        for (int ix = 0; ix < nxt; ix++) {
            // Wave velocity and stress tensor arrays
            vx[iz][ix] = 0.0;
            vz[iz][ix] = 0.0;
            sxx[iz][ix] = 0.0;
            szx[iz][ix] = 0.0;
            szz[iz][ix] = 0.0;
        }
    }
    // std::cout << "Reached Here1.5" << "\n";
    if (fwinv) {
        const int nft = 1 + (nt - 1) / fwi_dt;
        const int nfz = 1 + (fwi_z2 - fwi_z1) / fwi_dz;
        const int nfx = 1 + (fwi_x2 - fwi_x1) / fwi_dx;



        for (int it = 0; it < nft; it++) {
            for (int iz = 0; iz < nfz; iz++) {
                for (int ix = 0; ix < nfx; ix++) {

                    // FWI storage arrays
                    fwi_vx[it][iz][ix] = 0.0;
                    fwi_vz[it][iz][ix] = 0.0;
                    fwi_sxx[it][iz][ix] = 0.0;
                    fwi_szx[it][iz][ix] = 0.0;
                    fwi_szz[it][iz][ix] = 0.0;

                }
            }
        }

    }

    // Start of time loop
    isnap = 0;
    //**********************************Memory Copy**************************************
    clock_t start = 0, end = 0;
    double start1 = 0;
    double cpu_time_used;

    real_sim* d_a; real_sim* d_b; real_sim* d_K; real_sim* d_a_half; real_sim* d_b_half; real_sim* d_K_half;

    int size_1d = 2 * (npml + 1);

    gpuErrchk(hipMalloc((void**)&d_a, size_1d * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_b, size_1d * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_K, size_1d * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_a_half, size_1d * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_b_half, size_1d * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_K_half, size_1d * sizeof(real_sim)));


    real_sim* d_vx; real_sim* d_vz; real_sim* d_sxx;
    real_sim* d_szx; real_sim* d_szz; real_sim* d_lam;  real_sim* d_mu;
    real_sim* d_mu_zx; real_sim* d_rho_zp; real_sim* d_rho_xp;
    real_sim* d_mem_vx_x; real_sim* d_mem_vx_z; real_sim* d_mem_vz_x; real_sim* d_mem_vz_z;
    real_sim* d_mem_sxx_x; real_sim* d_mem_szx_x; real_sim* d_mem_szz_z; real_sim* d_mem_szx_z;

    real_sim size = nzt * nxt;

    gpuErrchk(hipMalloc((void**)&d_vx, size * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_vz, size * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_sxx, size * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_szx, size * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_szz, size * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_lam, size * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_mu, size * sizeof(real_sim)));
    size = (nzt - 1) * (nxt - 1);
    gpuErrchk(hipMalloc((void**)&d_mu_zx, size * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_rho_zp, size * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_rho_xp, size * sizeof(real_sim)));
    size = nzt * 2 * (npml + 1);
    gpuErrchk(hipMalloc((void**)&d_mem_vx_x, size * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_mem_vx_z, size * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_mem_vz_x, size * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_mem_vz_z, size * sizeof(real_sim)));

    gpuErrchk(hipMalloc((void**)&d_mem_sxx_x, size * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_mem_szx_x, size * sizeof(real_sim)));
    size = nxt * 2 * (npml + 1);
    gpuErrchk(hipMalloc((void**)&d_mem_szz_z, size * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_mem_szx_z, size * sizeof(real_sim)));

    size = nzt * nxt;
 
          // updating velocity tensors

    gpuErrchk(hipMemcpy(d_mem_vx_x, mem_vx_x[0], nzt * 2 * (npml + 1) * sizeof(real_sim), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_mem_vx_z, mem_vx_z[0], nzt * 2 * (npml + 1) * sizeof(real_sim), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_mem_vz_x, mem_vz_x[0], nzt * 2 * (npml + 1) * sizeof(real_sim), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_mem_vz_z, mem_vz_z[0], nzt * 2 * (npml + 1) * sizeof(real_sim), hipMemcpyHostToDevice));

    /* gpuErrchk(hipMemcpy(d_sxx, sxx[0], size * sizeof(real_sim), hipMemcpyHostToDevice));
     gpuErrchk(hipMemcpy(d_szx, szx[0], size * sizeof(real_sim), hipMemcpyHostToDevice));
     gpuErrchk(hipMemcpy(d_szz, szz[0], size * sizeof(real_sim), hipMemcpyHostToDevice));*/
    gpuErrchk(hipMemcpy(d_lam, lam[0], size * sizeof(real_sim), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_mu, mu[0], size * sizeof(real_sim), hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(d_mu_zx, mu_zx[0], (nzt - 1) * (nxt - 1) * sizeof(real_sim), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_rho_zp, rho_zp[0], (nzt - 1) * (nxt - 1) * sizeof(real_sim), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_rho_xp, rho_xp[0], (nzt - 1) * (nxt - 1) * sizeof(real_sim), hipMemcpyHostToDevice));


    gpuErrchk(hipMemcpy(d_mem_sxx_x, mem_sxx_x[0], nzt * 2 * (npml + 1) * sizeof(real_sim), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_mem_szx_x, mem_szx_x[0], nzt * 2 * (npml + 1) * sizeof(real_sim), hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(d_mem_szz_z, mem_szz_z[0], nxt * 2 * (npml + 1) * sizeof(real_sim), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_mem_szx_z, mem_szx_z[0], nxt * 2 * (npml + 1) * sizeof(real_sim), hipMemcpyHostToDevice));


    gpuErrchk(hipMemcpy(d_a, a, size_1d * sizeof(real_sim), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_b, b, size_1d * sizeof(real_sim), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_K, K, size_1d * sizeof(real_sim), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_a_half, a_half, size_1d * sizeof(real_sim), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_b_half, b_half, size_1d * sizeof(real_sim), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_K_half, K_half, size_1d * sizeof(real_sim), hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(d_vx, vx[0], size * sizeof(real_sim), hipMemcpyHostToDevice));
   
   

    int box1 = 16, box2 = 16;
    dim3 threadsPerBlock(box1, box2);
    dim3 blocksPerGrid((nz2 + box1 - 1) / box1, (nx2 + box2 - 1) / box2);


    real_sim* d_fwi_vx;
    real_sim* d_fwi_vz;
    real_sim* d_fwi_sxx;
    real_sim* d_fwi_szx;
    real_sim* d_fwi_szz;

    const int nft = 1 + (nt - 1) / fwi_dt;
    const int nfz = 1 + (fwi_z2 - fwi_z1) / fwi_dz;
    const int nfx = 1 + (fwi_x2 - fwi_x1) / fwi_dx;

    gpuErrchk(hipMalloc((void**)&d_fwi_vx, (nft * nfz * nfx) * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_fwi_vz, (nft * nfz * nfx) * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_fwi_sxx, (nft * nfz * nfx) * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_fwi_szx, (nft * nfz * nfx) * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_fwi_szz, (nft * nfz * nfx) * sizeof(real_sim)));

    gpuErrchk(hipMemcpy(d_fwi_vx, fwi_vx[0][0], (nft * nfz * nfx) * sizeof(real_sim), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_fwi_vz, fwi_vz[0][0], (nft * nfz * nfx) * sizeof(real_sim), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_fwi_sxx, fwi_sxx[0][0], (nft * nfz * nfx) * sizeof(real_sim), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_fwi_szx, fwi_szx[0][0], (nft * nfz * nfx) * sizeof(real_sim), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_fwi_szz, fwi_szz[0][0], (nft * nfz * nfx) * sizeof(real_sim), hipMemcpyHostToDevice));

   
    //**************************************************************************************************


    for (int it = 0; it < nt; it++) {
        start = clock();

        // ---------------------------------------------------------
        // Storing velocity  & stress tensors for gradient calculations for full waveform inversion
        if (fwinv && !(it % fwi_dt)) {
            tf = it / fwi_dt; // t index for fwi gradient storage
            //std::cout<<"fwi time: " << it << ", forward simulation" << std::endl;

            kernel_zero << < blocksPerGrid, threadsPerBlock >> > (tf, nzt, fwi_z1, fwi_z2, fwi_x1, fwi_x2,
                fwi_dz, fwi_dx, nft, nfz, nfx, d_fwi_sxx, d_fwi_szx, d_fwi_szz, d_fwi_vx,
                d_fwi_vz, d_sxx, d_szx, d_szz, d_vx, d_vz);

        }

        //******************************************GPU****************************************


        gpuErrchk(hipMemcpy(d_vz, vz[0], size * sizeof(real_sim), hipMemcpyHostToDevice));

        // Calculate spatial velocity derivatives
        kernel_one << < blocksPerGrid, threadsPerBlock >> > (ishot, nt, nzt, nxt, fpad, ppad, dt, dx, dz,
            fdorder, d_vx, d_vz, d_sxx,
            d_szx, d_szz, d_lam, d_mu,
            d_mu_zx, d_rho_zp, d_rho_xp, npml,
            d_a, d_b, d_K, d_a_half, d_b_half, d_K_half,
            d_mem_vx_x, d_mem_vx_z, d_mem_vz_x, d_mem_vz_z,
            d_mem_sxx_x, d_mem_szx_x, d_mem_szz_z, d_mem_szx_z, fsurf);
      
        gpuErrchk(hipPeekAtLastError());

        // compute spatial stress derivatives
        kernel_two << < blocksPerGrid, threadsPerBlock >> > (ishot, nt, nzt, nxt, fpad, ppad, dt, dx, dz,
            fdorder, d_vx, d_vz, d_sxx,
            d_szx, d_szz, d_lam, d_mu,
            d_mu_zx, d_rho_zp, d_rho_xp, npml,
            d_a, d_b, d_K, d_a_half, d_b_half, d_K_half,
            d_mem_vx_x, d_mem_vx_z, d_mem_vz_x, d_mem_vz_z,
            d_mem_sxx_x, d_mem_szx_x, d_mem_szz_z, d_mem_szx_z, fsurf);
        gpuErrchk(hipDeviceSynchronize());
        gpuErrchk(hipPeekAtLastError());

        gpuErrchk(hipMemcpy(vz[0], d_vz, size * sizeof(real_sim), hipMemcpyDeviceToHost));

        if (fsurf) { // Mirroring stresses for free surface condition

            kernel_Thri << < blocksPerGrid, threadsPerBlock >> > (nx1, nx2, fpad, nzt, d_szx, d_szz);

        }


        //****************************************************************************************************************




                // Adding Velocity update related sources

                //--------------------------
        for (int is = 0; is < nsrc; is++) {

            if (source_to_fire_shot[is] == ishot) {
                switch (src_comp[is]) {// defines the signal type
                case(2): // vz component only
                    vz[src_z[is]][src_x[is]] += src_signal[is][it];
                }
            }
        }

 

        // Recording the signals to the receivers
        for (int ir = 0; ir <= 0 /*nrec*/; ir++) {

            if (receiver_to_record_shot[ir] == ishot) {

                switch (rec_comp[ir]) {// defines the signal type
                case(2): // vz component only
                    rec_signal[ir][it] = vz[rec_z[ir]][rec_x[ir]];
                }

            }
        }


        // Printing out AASCII data for snap intervals
        if (!(it % snap_interval || it == 0)) {
            std::cout << "Time step " << it << " of " << nt << " in forward kernel." << std::endl;
            /*  outFile.open("./io/snap_data/vz_snap" + std::to_string(isnap) + ".csv");

              for (int j = 0; j < nzt; j++) {
                  for (int i = 0; i < nxt; i++) {
                      outFile << vz[j][i] << ", ";
                  }
                  outFile << std::endl;
              }
              outFile.close();*/
            isnap++;


        }



        
    } // end of time loop

    //******************************memcopy back to host************************************

    gpuErrchk(hipMemcpy(fwi_vx[0][0], d_fwi_vx, (nft * nfz * nfx) * sizeof(real_sim), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(fwi_vz[0][0], d_fwi_vz, (nft * nfz * nfx) * sizeof(real_sim), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(fwi_sxx[0][0], d_fwi_sxx, (nft * nfz * nfx) * sizeof(real_sim), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(fwi_szx[0][0], d_fwi_szx, (nft * nfz * nfx) * sizeof(real_sim), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(fwi_szz[0][0], d_fwi_szz, (nft * nfz * nfx) * sizeof(real_sim), hipMemcpyDeviceToHost));

    gpuErrchk(hipMemcpy(vx[0], d_vx, size * sizeof(real_sim), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(sxx[0], d_sxx, size * sizeof(real_sim), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(szx[0], d_szx, size * sizeof(real_sim), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(szz[0], d_szz, size * sizeof(real_sim), hipMemcpyDeviceToHost));

    gpuErrchk(hipMemcpy(lam[0], d_lam, size * sizeof(real_sim), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(mu[0], d_mu, size * sizeof(real_sim), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(mu_zx[0], d_mu_zx, (nzt - 1) * (nxt - 1) * sizeof(real_sim), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(rho_zp[0], d_rho_zp, (nzt - 1) * (nxt - 1) * sizeof(real_sim), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(rho_xp[0], d_rho_xp, (nzt - 1) * (nxt - 1) * sizeof(real_sim), hipMemcpyDeviceToHost));

    gpuErrchk(hipMemcpy(mem_vx_x[0], d_mem_vx_x, nzt * 2 * (npml + 1) * sizeof(real_sim), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(mem_vx_z[0], d_mem_vx_z, nzt * 2 * (npml + 1) * sizeof(real_sim), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(mem_vz_x[0], d_mem_vz_x, nzt * 2 * (npml + 1) * sizeof(real_sim), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(mem_vz_z[0], d_mem_vz_z, nzt * 2 * (npml + 1) * sizeof(real_sim), hipMemcpyDeviceToHost));

    gpuErrchk(hipMemcpy(mem_sxx_x[0], d_mem_sxx_x, nzt * 2 * (npml + 1) * sizeof(real_sim), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(mem_szx_x[0], d_mem_szx_x, nzt * 2 * (npml + 1) * sizeof(real_sim), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(mem_szz_z[0], d_mem_szz_z, nxt * 2 * (npml + 1) * sizeof(real_sim), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(mem_szx_z[0], d_mem_szx_z, nxt * 2 * (npml + 1) * sizeof(real_sim), hipMemcpyDeviceToHost))
    gpuErrchk(hipDeviceReset());



}