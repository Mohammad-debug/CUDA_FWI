#include "hip/hip_runtime.h"
//forward_kernel_PSV.cpp

/*
* Created by: Min Basnet
* 2020.April.16
* Kathmandu, Nepal
*/

#include <iostream>
#include <fstream>
#include <cmath>
#include <vector>

#include "globvar.cuh"
#include "util.cu"
#include "fd_cpml.cuh"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}
__global__ void kernel_I(int tf, int fwi_dt, int dt, int nzt, int fwi_z1, int fwi_z2, int fwi_x1, int fwi_x2,
    int fwi_dz, int fwi_dx, int nft, int nfz, int nfx,
    real_sim* fwi_sxx, real_sim* fwi_szx, real_sim* fwi_szz, real_sim* fwi_vx, real_sim*
    fwi_vz, real_sim* sxx, real_sim* szx, real_sim* szz, real_sim* vx, real_sim* vz, real_sim* mu,
    real_sim* lam, real_sim* grad_lam, real_sim* grad_mu, real_sim* grad_rho) {

    int iz = blockIdx.x * blockDim.x + threadIdx.x;
    int ix = blockIdx.y * blockDim.y + threadIdx.y;
    real_sim s1, s2, s3, s4;
    if (iz >= fwi_z1 && iz < fwi_z2 && iz % fwi_dz == fwi_z1) { // storing only a portion and with grid inteval
        int zf = (iz - fwi_z1) / fwi_dz; // z index for fwi gradient storage

        if (ix >= fwi_x1 && ix < fwi_x2 && ix % fwi_dx == fwi_x1) {
            int xf = (ix - fwi_x1) / fwi_dx; // x index for fwi gradient storage
            int offset = tf * nft * nfz + zf * nfz + xf;


            s1 = (fwi_sxx[offset] + fwi_szz[offset]) * (sxx[iz * nzt + ix] + szz[iz * nzt + ix])
                * 0.25 / ((lam[iz * nzt + ix] + mu[iz * nzt + ix]) * (lam[iz * nzt + ix] + mu[iz * nzt + ix]));

            s2 = (fwi_sxx[offset] - fwi_szz[offset]) * (sxx[iz * nzt + ix] - szz[iz * nzt + ix])
                / (mu[iz * nzt + ix] * mu[iz * nzt + ix]);

            s3 = (fwi_szx[offset] * szx[iz * nzt + ix]) / (mu[iz * nzt + ix] * mu[iz * nzt + ix]);

            // The time derivatives of the velocity may have to be computed differently
            s4 = vx[iz * nzt + ix] * fwi_vx[offset] + vz[iz * nzt + ix] * fwi_vz[offset];

            grad_lam[zf * nfz + xf] += fwi_dt * dt * s1;
            grad_mu[zf * nfz + xf] += fwi_dt * dt * (s3 + s1 + s2);
            grad_rho[zf * nfz + xf] += fwi_dt * dt * s4;


        }
    }
    else {
        return;
    }
}

__global__ void kernel_II(int ishot, int nt, int nzt, int nxt, int fpad, int ppad, real_sim dt, real_sim dx, real_sim dz,
    int fdorder, real_sim* vx, real_sim* vz, real_sim* sxx,
    real_sim* szx, real_sim* szz, real_sim* lam, real_sim* mu,
    real_sim* mu_zx, real_sim* rho_zp, real_sim* rho_xp, int npml,
    real_sim* a, real_sim* b, real_sim* K, real_sim* a_half, real_sim* b_half, real_sim* K_half,
    real_sim* mem_vx_x, real_sim* mem_vx_z, real_sim* mem_vz_x, real_sim* mem_vz_z,
    real_sim* mem_sxx_x, real_sim* mem_szx_x, real_sim* mem_szz_z, real_sim* mem_szx_z, bool fsurf) {

    //**********************************************************************************
    real_sim sxx_x, szx_x, szx_z, szz_z; // spatial stress derivatives
    real_sim vx_x, vx_z, vz_x, vz_z; // spatial velocity derivatives
    int nz1, nz2, nx1, nx2; // The computational grid boundaries
    int px, pz; // index for PML arrys
    int isnap; // to take snapshots for data storage

    int tf, zf, xf; // Index parameters for fwi data storage

    real_sim hc[2] = { 1.0, 1.0 };

    // Initial calculation of indices
    //---------------------------------------------
    nz1 = fpad; nz2 = nzt - fpad; nx1 = fpad; nx2 = nxt - fpad; // index variables

    // index to map PML at the right or positive end
    int pnx, pnz;
    pnx = nxt - 2 * ppad + fpad - 1; // nx + ppad + npml + 1 and nx = nxt - 2*ppad

    if (fsurf) {
        pnz = nzt - 2 * ppad - 1; // nz + ppad + npml + 1 and nz = nzt - ppad - fpad
    }
    else {
        pnz = nzt - 2 * ppad + fpad - 1; // nz + ppad + npml + 1 and nz = nzt - 2*ppad
    }

    real_sim dxi = 1.0 / dx; real_sim dzi = 1.0 / dz; // inverse of dx and dz
//**********************************************************************************************************


    //int k = blockIdx.x * blockDim.x + threadIdx.x;
    //int ix = k % nx2;
    //int iz = k % nz2;
    int iz = blockIdx.x * blockDim.x + threadIdx.x;
    int ix = blockIdx.y * blockDim.y + threadIdx.y;

    //if (threadIdx.x == 0 && blockIdx.x == 0)
    //    printf("Debug Params : . ..");

    ////***********************************************
    //switch (fdorder) {
    //case(2): {

    //    // Calculate spatial velocity derivatives


    switch (fdorder) {
    case(2):
        if (ix < nx2 && ix >= nx1 && iz >= nz1 && iz < nz2)
        {

            // Calculate spatial velocity derivatives
            vx_x = dxi * hc[1] * (vx[iz * nzt + ix] - vx[iz * nzt + (ix - 1)]);
            vz_x = dxi * hc[1] * (vz[iz * nzt + (ix + 1)] - vz[iz * nzt + ix]);
            vx_z = dzi * hc[1] * (vx[(iz + 1) * nzt + ix] - vx[iz * nzt + ix]);
            vz_z = dzi * hc[1] * (vz[iz * nzt + ix] - vz[(iz - 1) * nzt + ix]);
           
    // ---------------------------------------------------
    // CPML layers for stress tensor kernel
    // ---------------------------------------------------



            if (npml > 0) {

                if (ix >= fpad && ix <= ppad)
                { // left CPML
                  // Mapping the static CPML and memory variables to
                    px = ix - fpad; // the memory array index
                    //std::cout << std::endl << "Fault1 " << ix << std::endl;


                    mem_vx_x[iz * nzt + px] = b[px] * mem_vx_x[iz * nzt + px] + a[px] * vx_x;
                    mem_vz_x[iz * nzt + px] = b_half[px] * mem_vz_x[iz * nzt + px] + a_half[px] * vz_x;

                    vx_x = vx_x / K[px] + mem_vx_x[iz * nzt + px];
                    vz_x = vz_x / K_half[px] + mem_vz_x[iz * nzt + px];

                } // cpml left



                if (ix >= (nxt - ppad - 1) && ix < nxt - fpad) { // right CPML
                  // Mapping the static CPML and memory variables to

                    px = ix - pnx; // The PML factors index

                    mem_vx_x[iz * nzt + px] = b[px] * mem_vx_x[iz * nzt + px] + a[px] * vx_x;
                    mem_vz_x[iz * nzt + px] = b_half[px] * mem_vz_x[iz * nzt + px] + a_half[px] * vz_x;

                    vx_x = vx_x / K[px] + mem_vx_x[iz * nzt + px];
                    vz_x = vz_x / K_half[px] + mem_vz_x[iz * nzt + px];


                } // cpml right


                if (iz >= fpad && iz <= ppad && !fsurf) { // top CPML
                  // Mapping the static CPML and memory variables to
                    pz = iz - fpad; // the memory array index

                    mem_vz_z[pz * 2 * (npml + 1) + ix] = b[pz] * mem_vz_z[pz * 2 * (npml + 1) + ix] + a[pz] * vz_z;
                    mem_vx_z[pz * 2 * (npml + 1) + ix] = b_half[pz] * mem_vx_z[pz * 2 * (npml + 1) + ix] + a_half[pz] * vx_z;

                    vz_z = vz_z / K[pz] + mem_vz_z[pz * 2 * (npml + 1) + ix];
                    vx_z = vx_z / K_half[pz] + mem_vx_z[pz * 2 * (npml + 1) + ix];

                    //std::cout << pz<< ", ";

                } // cpml top



                if (iz >= (nzt - ppad - 1) && iz < nzt - fpad) { // bottom CPML
                  // Mapping the static CPML and memory variables to
                    pz = iz - pnz; // The PML factors index

                    mem_vz_z[pz * 2 * (npml + 1) + ix] = b[pz] * mem_vz_z[pz * 2 * (npml + 1) + ix] + a[pz] * vz_z;
                    mem_vx_z[pz * 2 * (npml + 1) + ix] = b_half[pz] * mem_vx_z[pz * 2 * (npml + 1) + ix] + a_half[pz] * vx_z;

                    vz_z = vz_z / K[pz] + mem_vz_z[pz * 2 * (npml + 1) + ix];
                    vx_z = vx_z / K_half[pz] + mem_vx_z[pz * 2 * (npml + 1) + ix];

                    //std::cout << pz<< ", ";

                } // cpml bottom

            }    // npml>0
            __syncthreads();
            //// --------------------------------------------------------------------------
            //// --------------------------------------------------------------------------

            ////// updating stresses

            szx[iz * nzt + ix] += dt * mu_zx[iz * (nzt - 1) + ix] * (vz_x + vx_z);
            sxx[iz * nzt + ix] += dt * (lam[iz * nzt + ix] * (vx_x + vz_z) + (2.0f * mu[iz * nzt + ix] * vx_x));
            szz[iz * nzt + ix] += dt * (lam[iz * nzt + ix] * (vx_x + vz_z) + (2.0f * mu[iz * nzt + ix] * vz_z));


            // -----------------------------------------
   // ----------------------------------------- 
   // Override stress for free surface implementation

            if (fsurf && iz == fpad) {
                // Free surface at z = 0 or nz = fpad

                // Denise manual  page 13
                szz[fpad * nzt + ix] = 0.0;
                szx[fpad * nzt + ix] = 0.0;
                sxx[fpad * nzt + ix] = 4.0 * dt * vx_x * (lam[fpad * nzt + ix] * mu[fpad * nzt + ix] + mu[fpad * nzt + ix] * mu[fpad * nzt + ix])
                    / (lam[fpad * nzt + ix] + 2.0 * mu[fpad * nzt + ix]);

            }

        }
        else {
            return;
        }

    default:
        //  std::cout << "FDORDER = " << fdorder << npml << std::endl;
        //  std::cout << "FD order mismatch. <SIMULATION ABORTED>" << std::endl;
          //exit(0);
       // printf("FD order mismatch. <SIMULATION ABORTED>\n");
    } // end of switch


}

__global__ void kernel_III(int ishot, int nt, int nzt, int nxt, int fpad, int ppad, real_sim dt, real_sim dx, real_sim dz,
    int fdorder, real_sim* vx, real_sim* vz, real_sim* sxx,
    real_sim* szx, real_sim* szz, real_sim* lam, real_sim* mu,
    real_sim* mu_zx, real_sim* rho_zp, real_sim* rho_xp, int npml,
    real_sim* a, real_sim* b, real_sim* K, real_sim* a_half, real_sim* b_half, real_sim* K_half,
    real_sim* mem_vx_x, real_sim* mem_vx_z, real_sim* mem_vz_x, real_sim* mem_vz_z,
    real_sim* mem_sxx_x, real_sim* mem_szx_x, real_sim* mem_szz_z, real_sim* mem_szx_z, bool fsurf) {


    real_sim sxx_x, szx_x, szx_z, szz_z; // spatial stress derivatives
    real_sim vx_x, vx_z, vz_x, vz_z; // spatial velocity derivatives
    int nz1, nz2, nx1, nx2; // The computational grid boundaries
    int px, pz; // index for PML arrys
    int isnap; // to take snapshots for data storage

    int tf, zf, xf; // Index parameters for fwi data storage


    real_sim hc[2] = { 1.0, 1.0 };

    // Initial calculation of indices
    //---------------------------------------------
    nz1 = fpad; nz2 = nzt - fpad; nx1 = fpad; nx2 = nxt - fpad; // index variables

    // index to map PML at the right or positive end
    int pnx, pnz;
    pnx = nxt - 2 * ppad + fpad - 1; // nx + ppad + npml + 1 and nx = nxt - 2*ppad

    if (fsurf) {
        pnz = nzt - 2 * ppad - 1; // nz + ppad + npml + 1 and nz = nzt - ppad - fpad
    }
    else {
        pnz = nzt - 2 * ppad + fpad - 1; // nz + ppad + npml + 1 and nz = nzt - 2*ppad
    }

    real_sim dxi = 1.0 / dx; real_sim dzi = 1.0 / dz; // inverse of dx and dz


    //*************************************************

    int iz = blockIdx.x * blockDim.x + threadIdx.x;
    int ix = blockIdx.y * blockDim.y + threadIdx.y;


    switch (fdorder) {
    case(2):
        if (ix < nx2 && ix >= nx1 && iz >= nz1 && iz < nz2)
        {
            // printf("abc");
            // compute spatial stress derivatives
            sxx_x = dxi * hc[1] * (sxx[iz * nzt + ix + 1] - sxx[iz * nzt + ix]);
            szx_z = dxi * hc[1] * (szx[iz * nzt + ix] - szx[(iz - 1) * nzt + ix]);

            szx_x = dzi * hc[1] * (szx[iz * nzt + ix] - szx[iz * nzt + ix - 1]);
            szz_z = dzi * hc[1] * (szz[(iz + 1) * nzt + ix] - szz[iz * nzt + ix]);


            // ---------------------------------------------------
            // CPML layers for particle velocity kernel
            // ---------------------------------------------------

            if (npml > 0) {

                if (ix >= fpad && ix < ppad) { // left CPML
                  // Mapping the static CPML and memory variables to
                    px = ix - fpad; // the memory array index

                    mem_sxx_x[iz * nzt + px] = b[px] * mem_sxx_x[iz * nzt + px] + a[px] * sxx_x;
                    mem_szx_x[iz * nzt + px] = b_half[px] * mem_szx_x[iz * nzt + px] + a_half[px] * szx_x;

                    sxx_x = sxx_x / K[px] + mem_sxx_x[iz * nzt + px];
                    szx_x = szx_x / K_half[px] + mem_szx_x[iz * nzt + px];

                } // cpml left

                if (ix >= (nxt - ppad - 1) && ix < (nxt - fpad)) { // right CPML
                  // Mapping the static CPML and memory variables to
                    px = ix - pnx; // The PML factors index

                    mem_sxx_x[iz * nzt + px] = b[px] * mem_sxx_x[iz * nzt + px] + a[px] * sxx_x;
                    mem_szx_x[iz * nzt + px] = b_half[px] * mem_szx_x[iz * nzt + px] + a_half[px] * szx_x;

                    sxx_x = sxx_x / K[px] + mem_sxx_x[iz * nzt + px];
                    szx_x = szx_x / K_half[px] + mem_szx_x[iz * nzt + px];

                } // cpml right


                if (iz >= fpad && iz < ppad && !fsurf) { // top CPML
                  // Mapping the static CPML and memory variables to
                    pz = iz - fpad; // the memory array index

                    mem_szz_z[pz * 2 * (npml + 1) + ix] = b[pz] * mem_szz_z[pz * 2 * (npml + 1) + ix] + a[pz] * szz_z;
                    mem_szx_z[pz * 2 * (npml + 1) + ix] = b_half[pz] * mem_szx_z[pz * 2 * (npml + 1) + ix] + a_half[pz] * szx_z;

                    szz_z = szz_z / K[pz] + mem_szz_z[pz * 2 * (npml + 1) + ix];
                    szx_z = szx_z / K_half[pz] + mem_szx_z[pz * 2 * (npml + 1) + ix];

                } // cpml top

                if (iz >= (nzt - ppad - 1) && iz < nzt - fpad) { // bottom CPML
                  // Mapping the static CPML and memory variables to
                    pz = iz - pnz; // The PML factors index

                    mem_szz_z[pz * 2 * (npml + 1) + ix] = b[pz] * mem_szz_z[pz * 2 * (npml + 1) + ix] + a[pz] * szz_z;
                    mem_szx_z[pz * 2 * (npml + 1) + ix] = b_half[pz] * mem_szx_z[pz * 2 * (npml + 1) + ix] + a_half[pz] * szx_z;

                    szz_z = szz_z / K[pz] + mem_szz_z[pz * 2 * (npml + 1) + ix];
                    szx_z = szx_z / K_half[pz] + mem_szx_z[pz * 2 * (npml + 1) + ix];

                } // cpml bottom
                __syncthreads();
            } // npml>0

            // update particle velocities
            vx[iz * nzt + ix] += dt * rho_xp[iz * (nzt - 1) + ix] * (sxx_x + szx_z);
            vz[iz * nzt + ix] += dt * rho_zp[iz * (nzt - 1) + ix] * (szx_x + szz_z);


        }
        else { return; }

        //  break;

    default:
        //  std::cout << "FDORDER = " << fdorder << npml << std::endl;
        //  std::cout << "FD order mismatch. <SIMULATION ABORTED>" << std::endl;
          //exit(0);
       // printf("FD order mismatch. <SIMULATION ABORTED>\n");
    } // end of switch


}

__global__ void kernel_IV(int nx1, int nx2, int fpad, int nzt, real_sim* szx, real_sim* szz) {
    int iz = blockIdx.x * blockDim.x + threadIdx.x;
    int ix = blockIdx.y * blockDim.y + threadIdx.y;
    int sz = 1;
    if (ix >= nx1 && ix < nx2 && sz >= 1 && sz <= fpad) {
        // mirroring 
        szx[(fpad - sz) * nzt + ix] = -szx[(fpad + sz) * nzt + ix];
        szz[(fpad - sz) * nzt + ix] = -szz[(fpad + sz) * nzt + ix];

    }

}




void adjoint_kernel_PSV(int ishot, // shot index
                        // Time and space grid arguments
    int nt, int nzt, int nxt, int fpad, int ppad,
    real_sim dt, real_sim dx, real_sim dz, int snap_interval, bool fsurf,
    // computationsl arguments
    real_sim* hc, int fdorder,
    // Wave arguments
    real_sim** vx, real_sim** vz,
    real_sim** sxx, real_sim** szx, real_sim** szz,
    // Medium arguments
    real_sim** lam, real_sim** mu,
    real_sim** mu_zx, real_sim** rho_zp, real_sim** rho_xp,
    //PML arguments
    int npml, real_sim* a, real_sim* b, real_sim* K,
    real_sim* a_half, real_sim* b_half, real_sim* K_half,
    // PML memory arrays
    real_sim** mem_vx_x, real_sim** mem_vx_z, real_sim** mem_vz_x, real_sim** mem_vz_z,
    real_sim** mem_sxx_x, real_sim** mem_szx_x, real_sim** mem_szz_z, real_sim** mem_szx_z,
    // Source arguments
    int nsrc, ivec src_x, ivec src_z, ivec src_comp,
    real_sim** src_signal, ivec source_to_fire_shot,
    // FWI arguments
    bool fwinv, int fwi_dt, int fwi_dx, int fwi_dz,
    int fwi_x1, int fwi_x2, int fwi_z1, int fwi_z2,
    real_sim*** fwi_vx, real_sim*** fwi_vz, real_sim*** fwi_sxx,
    real_sim*** fwi_szx, real_sim*** fwi_szz,
    // Gradient of the materials
    real_sim** grad_lam, real_sim** grad_mu, real_sim** grad_rho) {

    //const bool fwi = 1;
    // int nt = number of timesteps
    // int nz1, nz2, nx1, nx2 = start and end grids along z and x directions
    // int dt, dx, dz = grid spacing in time and space
    // int* hc = holberg coefficients
    // real_sim **&vx, **&vz, **&sxx, **&szx, **&szz, // wave parameters (particle velocity and stresses)
    // real_sim **&lam, **&mu, **&mu_zx, **&rho_zp, **&rho_xp // medium parameters (lam�'s parameters')

    // real_sim *a, *b, *K;// CPML parameters
    // real_sim *a_half, *b_half, *K_half // CPML interpolated parameters
    // real_sim ** mem_vx_x, ** mem_vx_z, ** mem_vz_x, ** mem_vz_z; // PML velocity derivative memory
    // real_sim **&mem_sxx_x, **&mem_szx_x, **&mem_szz_z, real_sim **&mem_szx_z // PML stress derivative memory
    // bool fsurf :: free surface on the top


    // Source arguments
    // int nsrc = number of sources
    // int **src_loc = grid location of source + source parameter type for eg exploxive, vz only etc
    // real_sim ** src_signal = signal values for the sources

    real_sim sxx_x, szx_x, szx_z, szz_z; // spatial stress derivatives
    real_sim vx_x, vx_z, vz_x, vz_z; // spatial velocity derivatives
    int nz1, nz2, nx1, nx2; // The computational grid boundaries
    int px, pz; // index for PML arrys
    int isnap; // to take snapshots for data storage

    int tf, zf, xf; // Index parameters for fwi data storage
    real_sim s1, s2, s3, s4; // Intermediate variables for gradient calculation

    std::ofstream outFile; // file to print vz arrays


    // Initial calculation of indices
    //---------------------------------------------
    nz1 = fpad; nz2 = nzt - fpad; nx1 = fpad; nx2 = nxt - fpad; // index variables

    // index to map PML at the right or positive end
    int pnx, pnz;
    pnx = nxt - 2 * ppad + fpad - 1; // nx + ppad + npml + 1 and nx = nxt - 2*ppad

    if (fsurf) {
        pnz = nzt - 2 * ppad - 1; // nz + ppad + npml + 1 and nz = nzt - ppad - fpad
    }
    else {
        pnz = nzt - 2 * ppad + fpad - 1; // nz + ppad + npml + 1 and nz = nzt - 2*ppad
    }

    real_sim dxi = 1.0 / dx; real_sim dzi = 1.0 / dz; // inverse of dx and dz

    // -----------------------------------------------------------


    // Reset kernels
    // -----------------------------------------------------
    // Stress and velocity kernels
    for (int iz = 0; iz < nzt; iz++) {
        for (int ix = 0; ix < nxt; ix++) {
            // Wave velocity and stress tensor arrays
            vx[iz][ix] = 0.0;
            vz[iz][ix] = 0.0;
            sxx[iz][ix] = 0.0;
            szx[iz][ix] = 0.0;
            szz[iz][ix] = 0.0;
        }
    }

    // Gradient kernels
    //-----------------------------

    const int nft = 1 + (nt - 1) / fwi_dt;
    const int nfz = 1 + (fwi_z2 - fwi_z1) / fwi_dz;
    const int nfx = 1 + (fwi_x2 - fwi_x1) / fwi_dx;

    for (int iz = 0; iz < nfz; iz++) {
        for (int ix = 0; ix < nfx; ix++) {

            // Gradients of the material
            grad_lam[iz][ix] = 0.0;
            grad_mu[iz][ix] = 0.0;
            grad_rho[iz][ix] = 0.0;

        }
    }
    //*******************************************GPU***********************


    real_sim* d_a; real_sim* d_b; real_sim* d_K; real_sim* d_a_half; real_sim* d_b_half; real_sim* d_K_half;

    int size_1d = 2 * (npml + 1);

    gpuErrchk(hipMalloc((void**)&d_a, size_1d * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_b, size_1d * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_K, size_1d * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_a_half, size_1d * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_b_half, size_1d * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_K_half, size_1d * sizeof(real_sim)));


    real_sim* d_vx; real_sim* d_vz; real_sim* d_sxx;
    real_sim* d_szx; real_sim* d_szz; real_sim* d_lam;  real_sim* d_mu;
    real_sim* d_mu_zx; real_sim* d_rho_zp; real_sim* d_rho_xp;
    real_sim* d_mem_vx_x; real_sim* d_mem_vx_z; real_sim* d_mem_vz_x; real_sim* d_mem_vz_z;
    real_sim* d_mem_sxx_x; real_sim* d_mem_szx_x; real_sim* d_mem_szz_z; real_sim* d_mem_szx_z;

    real_sim size = nzt * nxt;

    gpuErrchk(hipMalloc((void**)&d_vx, size * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_vz, size * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_sxx, size * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_szx, size * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_szz, size * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_lam, size * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_mu, size * sizeof(real_sim)));
    size = (nzt - 1) * (nxt - 1);
    gpuErrchk(hipMalloc((void**)&d_mu_zx, size * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_rho_zp, size * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_rho_xp, size * sizeof(real_sim)));
    size = nzt * 2 * (npml + 1);
    gpuErrchk(hipMalloc((void**)&d_mem_vx_x, size * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_mem_vx_z, size * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_mem_vz_x, size * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_mem_vz_z, size * sizeof(real_sim)));

    gpuErrchk(hipMalloc((void**)&d_mem_sxx_x, size * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_mem_szx_x, size * sizeof(real_sim)));
    size = nxt * 2 * (npml + 1);
    gpuErrchk(hipMalloc((void**)&d_mem_szz_z, size * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_mem_szx_z, size * sizeof(real_sim)));

    size = nzt * nxt;
    

     //*****************
          // updating velocity tensors

    gpuErrchk(hipMemcpy(d_mem_vx_x, mem_vx_x[0], nzt * 2 * (npml + 1) * sizeof(real_sim), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_mem_vx_z, mem_vx_z[0], nzt * 2 * (npml + 1) * sizeof(real_sim), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_mem_vz_x, mem_vz_x[0], nzt * 2 * (npml + 1) * sizeof(real_sim), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_mem_vz_z, mem_vz_z[0], nzt * 2 * (npml + 1) * sizeof(real_sim), hipMemcpyHostToDevice));

     gpuErrchk(hipMemcpy(d_sxx, sxx[0], size * sizeof(real_sim), hipMemcpyHostToDevice));
     gpuErrchk(hipMemcpy(d_szx, szx[0], size * sizeof(real_sim), hipMemcpyHostToDevice));
     gpuErrchk(hipMemcpy(d_szz, szz[0], size * sizeof(real_sim), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_lam, lam[0], size * sizeof(real_sim), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_mu, mu[0], size * sizeof(real_sim), hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(d_mu_zx, mu_zx[0], (nzt - 1) * (nxt - 1) * sizeof(real_sim), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_rho_zp, rho_zp[0], (nzt - 1) * (nxt - 1) * sizeof(real_sim), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_rho_xp, rho_xp[0], (nzt - 1) * (nxt - 1) * sizeof(real_sim), hipMemcpyHostToDevice));


    gpuErrchk(hipMemcpy(d_mem_sxx_x, mem_sxx_x[0], nzt * 2 * (npml + 1) * sizeof(real_sim), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_mem_szx_x, mem_szx_x[0], nzt * 2 * (npml + 1) * sizeof(real_sim), hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(d_mem_szz_z, mem_szz_z[0], nxt * 2 * (npml + 1) * sizeof(real_sim), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_mem_szx_z, mem_szx_z[0], nxt * 2 * (npml + 1) * sizeof(real_sim), hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(d_a, a, size_1d * sizeof(real_sim), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_b, b, size_1d * sizeof(real_sim), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_K, K, size_1d * sizeof(real_sim), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_a_half, a_half, size_1d * sizeof(real_sim), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_b_half, b_half, size_1d * sizeof(real_sim), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_K_half, K_half, size_1d * sizeof(real_sim), hipMemcpyHostToDevice));


    gpuErrchk(hipMemcpy(d_vx, vx[0], size * sizeof(real_sim), hipMemcpyHostToDevice));


    real_sim* d_fwi_vx;
    real_sim* d_fwi_vz;
    real_sim* d_fwi_sxx;
    real_sim* d_fwi_szx;
    real_sim* d_fwi_szz;

    real_sim* d_grad_lam;
    real_sim* d_grad_mu;
    real_sim* d_grad_rho;

    int size_grad = nfz * nfx;
    gpuErrchk(hipMalloc((void**)&d_grad_lam, size_grad * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_grad_mu, size_grad * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_grad_rho, size_grad * sizeof(real_sim)));

    gpuErrchk(hipMemcpy(d_grad_lam, grad_lam[0], size_grad * sizeof(real_sim), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_grad_mu, grad_mu[0], size_grad * sizeof(real_sim), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_grad_rho, grad_rho[0], size_grad * sizeof(real_sim), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc((void**)&d_fwi_vx, (nft * nfz * nfx) * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_fwi_vz, (nft * nfz * nfx) * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_fwi_sxx, (nft * nfz * nfx) * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_fwi_szx, (nft * nfz * nfx) * sizeof(real_sim)));
    gpuErrchk(hipMalloc((void**)&d_fwi_szz, (nft * nfz * nfx) * sizeof(real_sim)));

    gpuErrchk(hipMemcpy(d_fwi_vx, fwi_vx[0][0], (nft * nfz * nfx) * sizeof(real_sim), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_fwi_vz, fwi_vz[0][0], (nft * nfz * nfx) * sizeof(real_sim), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_fwi_sxx, fwi_sxx[0][0], (nft * nfz * nfx) * sizeof(real_sim), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_fwi_szx, fwi_szx[0][0], (nft * nfz * nfx) * sizeof(real_sim), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_fwi_szz, fwi_szz[0][0], (nft * nfz * nfx) * sizeof(real_sim), hipMemcpyHostToDevice));
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipPeekAtLastError());

    int box1 = 16, box2 = 16;
    dim3 threadsPerBlock(box1, box2);
    dim3 blocksPerGrid((nz2 + box1 - 1) / box1, (nx2 + box2 - 1) / box2);




    //************************************************************

    // Start of time loop
    isnap = 0;
    for (int it = nt - 1; it >= 0; it--) {

        // ---------------------------------------------------------
        // Computation of gradient kernels
        gpuErrchk(hipMemcpy(d_vz, vz[0], size * sizeof(real_sim), hipMemcpyHostToDevice));

        if (fwinv && !(it % fwi_dt)) {


            tf = it / fwi_dt; // t index for fwi gradient storage
            //std::cout<<"fwi time: " << it << ", adjoint simulation" << std::endl;
            kernel_I << < blocksPerGrid, threadsPerBlock >> > (tf, fwi_dt, dt, nzt, fwi_z1, fwi_z2, fwi_x1, fwi_x2,
                fwi_dz, fwi_dx, nft, nfz, nfx,
                d_fwi_sxx, d_fwi_szx, d_fwi_szz, d_fwi_vx,
                d_fwi_vz, d_sxx, d_szx, d_szz, d_vx, d_vz, d_mu,
                d_lam, d_grad_lam, d_grad_mu, d_grad_rho);

        }

        //****************************************************Kernrl calls GPU*************************************




        // Calculate spatial velocity derivatives
        kernel_II << < blocksPerGrid, threadsPerBlock >> > (ishot, nt, nzt, nxt, fpad, ppad, dt, dx, dz,
            fdorder, d_vx, d_vz, d_sxx,
            d_szx, d_szz, d_lam, d_mu,
            d_mu_zx, d_rho_zp, d_rho_xp, npml,
            d_a, d_b, d_K, d_a_half, d_b_half, d_K_half,
            d_mem_vx_x, d_mem_vx_z, d_mem_vz_x, d_mem_vz_z,
            d_mem_sxx_x, d_mem_szx_x, d_mem_szz_z, d_mem_szx_z, fsurf);
       
        gpuErrchk(hipPeekAtLastError());

        // compute spatial stress derivatives
        kernel_III << < blocksPerGrid, threadsPerBlock >> > (ishot, nt, nzt, nxt, fpad, ppad, dt, dx, dz,
            fdorder, d_vx, d_vz, d_sxx,
            d_szx, d_szz, d_lam, d_mu,
            d_mu_zx, d_rho_zp, d_rho_xp, npml,
            d_a, d_b, d_K, d_a_half, d_b_half, d_K_half,
            d_mem_vx_x, d_mem_vx_z, d_mem_vz_x, d_mem_vz_z,
            d_mem_sxx_x, d_mem_szx_x, d_mem_szz_z, d_mem_szx_z, fsurf);
        gpuErrchk(hipPeekAtLastError());

      


        if (fsurf) { // Mirroring stresses for free surface condition

            kernel_IV << < blocksPerGrid, threadsPerBlock >> > (nx1, nx2, fpad, nzt, d_szx, d_szz);
            gpuErrchk(hipPeekAtLastError());

        }
        gpuErrchk(hipDeviceSynchronize());
       

        gpuErrchk(hipMemcpy(vz[0], d_vz, size * sizeof(real_sim), hipMemcpyDeviceToHost));

        //****************************************************************************

                // Adding Velocity update related sources
                //----------------------------------------
        for (int is = 0; is <= 0 /*nsrc*/; is++) {

            if (source_to_fire_shot[is] == ishot) {
                switch (src_comp[is]) {// defines the signal type
                case(2): // vz component only
                    vz[src_z[is]][src_x[is]] += src_signal[is][it];
                }
            }
        }

        // ---------------------------------------

        // Printing out AASCII data for snap intervals
        if (!(it % snap_interval || it == 0)) {
            std::cout << "Time step " << it << " of " << nt << " in adjoint kernel." << std::endl;
            isnap++;
        }

    } // end of time loop

    gpuErrchk(hipMemcpy(grad_lam[0], d_grad_lam, size_grad * sizeof(real_sim), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(grad_mu[0], d_grad_mu, size_grad * sizeof(real_sim), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(grad_rho[0], d_grad_rho, size_grad * sizeof(real_sim), hipMemcpyDeviceToHost));

    //********************************************************************Memcpy back to host
    gpuErrchk(hipMemcpy(fwi_vx[0][0], d_fwi_vx, (nft * nfz * nfx) * sizeof(real_sim), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(fwi_vz[0][0], d_fwi_vz, (nft * nfz * nfx) * sizeof(real_sim), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(fwi_sxx[0][0], d_fwi_sxx, (nft * nfz * nfx) * sizeof(real_sim), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(fwi_szx[0][0], d_fwi_szx, (nft * nfz * nfx) * sizeof(real_sim), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(fwi_szz[0][0], d_fwi_szz, (nft * nfz * nfx) * sizeof(real_sim), hipMemcpyDeviceToHost));

    gpuErrchk(hipMemcpy(vx[0], d_vx, size * sizeof(real_sim), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(sxx[0], d_sxx, size * sizeof(real_sim), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(szx[0], d_szx, size * sizeof(real_sim), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(szz[0], d_szz, size * sizeof(real_sim), hipMemcpyDeviceToHost));

    gpuErrchk(hipMemcpy(lam[0], d_lam, size * sizeof(real_sim), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(mu[0], d_mu, size * sizeof(real_sim), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(mu_zx[0], d_mu_zx, (nzt - 1) * (nxt - 1) * sizeof(real_sim), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(rho_zp[0], d_rho_zp, (nzt - 1) * (nxt - 1) * sizeof(real_sim), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(rho_xp[0], d_rho_xp, (nzt - 1) * (nxt - 1) * sizeof(real_sim), hipMemcpyDeviceToHost));

    gpuErrchk(hipMemcpy(mem_vx_x[0], d_mem_vx_x, nzt * 2 * (npml + 1) * sizeof(real_sim), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(mem_vx_z[0], d_mem_vx_z, nzt * 2 * (npml + 1) * sizeof(real_sim), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(mem_vz_x[0], d_mem_vz_x, nzt * 2 * (npml + 1) * sizeof(real_sim), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(mem_vz_z[0], d_mem_vz_z, nzt * 2 * (npml + 1) * sizeof(real_sim), hipMemcpyDeviceToHost));


    gpuErrchk(hipMemcpy(mem_sxx_x[0], d_mem_sxx_x, nzt * 2 * (npml + 1) * sizeof(real_sim), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(mem_szx_x[0], d_mem_szx_x, nzt * 2 * (npml + 1) * sizeof(real_sim), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(mem_szz_z[0], d_mem_szz_z, nxt * 2 * (npml + 1) * sizeof(real_sim), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(mem_szx_z[0], d_mem_szx_z, nxt * 2 * (npml + 1) * sizeof(real_sim), hipMemcpyDeviceToHost));

    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipDeviceReset());


}
